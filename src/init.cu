#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include "../header/init.h"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "../header/struct.h"

#define MIN(i, j) (((i) < (j)) ? (i) : (j))
#define MAX(i, j) (((i) > (j)) ? (i) : (j))

void initMedium(Medium *med) {
  for (int mednum = 0; mednum < E_M_END; mednum++) {
    switch (mednum) {
      case E_AIR:
        med[mednum].rho = 1.205;  // 密度/////////////////////////////////////////////////////
        med[mednum].K = 1.422e5;  // 体積弾性率
        med[mednum].E = 0.;       // ヤング率
        med[mednum].G = 0.;       // 剛性率/////////////////////////////////////////////////////
        med[mednum].nu = 0.;     // ポアソン比
        med[mednum].ramda = med[mednum].K - 2. / 3. * med[mednum].G;  // 第1ラメ定数/////////////////////////////////////////////////////
        med[mednum].zeta = 5.;    //セル間摩擦係数/////////////////////////////////////////////////////
        med[mednum].gamma = 1.8e-5;   //第1粘性/////////////////////////////////////////////////////
        med[mednum].khi = 0.;         //第2粘性/////////////////////////////////////////////////////
        med[mednum].eta = 0.;
        med[mednum].omega = 0.;
        break;
      case E_CON:
        med[mednum].rho = 2400.;  // 密度/////////////////////////////////////////////////////
        med[mednum].E = 2.4e10;   // ヤング率
        med[mednum].nu = 0.2;     // ポアソン比
        med[mednum].G = med[mednum].E / 2. / (1. + med[mednum].nu);  // 剛性率////////////////////////////////////第2ラメ
        med[mednum].K = med[mednum].E / 3. / (1. - 2. * med[mednum].nu);  // 体積弾性率
        med[mednum].ramda = med[mednum].E * med[mednum].nu / (1. + med[mednum].nu) / (1. - 2. * med[mednum].nu);  // 第1ラメ定数//////////////
        med[mednum].zeta = 2.5e4;//セル間摩擦係数////////////////////////////////////////////////////////////////////////////////////
        med[mednum].eta = 0.005;//粘性定数算出係数(損失係数)
        med[mednum].omega = 2. * M_PI * 32.;//粘性定数算出係数(角周波数)
        med[mednum].gamma = med[mednum].eta * med[mednum].G / med[mednum].omega;//第1粘性定数//////////////////////////////////////
        med[mednum].khi = med[mednum].eta * med[mednum].ramda / med[mednum].omega;//第2粘性定数/////////////////////////////////////
        break;
      default:
        break;
    }
  }
}

void initCoord(Coord *co, int x, int y, int z) {
  co->x = x;
  co->y = y;
  co->z = z;
}

void initDiff(Diff *dif, Medium *med) {
  dif->dx = 0.005;
  dif->dy = 0.005;
  dif->dz = 0.005;
  double tmp;
  
  for(int i = E_AIR; i < E_M_END - 1; i++){
    tmp = MAX(sqrt((med[i].K + 4. / 3. * med[i].G) / med[i].rho),tmp);
  }
  printf("v = %lf\n", tmp);
  dif->dt = dif->dx / tmp / 100.;
}

void initPml(Pml *pml, Medium *med, Diff dif) {
  pml->ta = 4.;
  pml->fm = 3.574e4;
  double R = 1.e-20;
  double tmp,tmp_v;//max
  initCoord(&pml->pl1, 32, 32, 32);
  initCoord(&pml->pl2, 32, 32, 32);
  //計算領域内最高速度
  for(int i = E_AIR; i < E_M_END - 1; i++){
    tmp_v = MAX(sqrt((med[i].K + 4. / 3. * med[i].G) / med[i].rho),tmp);
  }
  //減衰係数最大値(PML層)
  for (int i = E_AIR + 1; i < E_M_END; i++) {
    tmp = tmp_v * (pml->ta + 1) / (2. * (double)pml->pl1.x * dif.dx) * log(1/R);
    pml->fm = MAX(tmp, pml->fm);
  }
}

void initRange(Range *ran, Coord region, Pml pml) {
  // initCoord(&ran->sr.Txx, x + pml.pl1.x + pml.pl2.x    , y + pml.pl1.y + pml.pl2.y    , z + pml.pl1.z + pml.pl2.z    );
  // initCoord(&ran->sr.Tyy, x + pml.pl1.x + pml.pl2.x    , y + pml.pl1.y + pml.pl2.y    , z + pml.pl1.z + pml.pl2.z    );
  // initCoord(&ran->sr.Tzz, x + pml.pl1.x + pml.pl2.x    , y + pml.pl1.y + pml.pl2.y    , z + pml.pl1.z + pml.pl2.z    );
  // initCoord(&ran->tr.Txy, x + pml.pl1.x + pml.pl2.x - 1, y + pml.pl1.y + pml.pl2.y - 1, z + pml.pl1.z + pml.pl2.z    );
  // initCoord(&ran->tr.Tyz, x + pml.pl1.x + pml.pl2.x    , y + pml.pl1.y + pml.pl2.y - 1, z + pml.pl1.z + pml.pl2.z - 1);
  // initCoord(&ran->tr.Tzx, x + pml.pl1.x + pml.pl2.x - 1, y + pml.pl1.y + pml.pl2.y    , z + pml.pl1.z + pml.pl2.z - 1);
  // initCoord(&ran->vr.Vx , x + pml.pl1.x + pml.pl2.x - 1, y + pml.pl1.y + pml.pl2.y    , z + pml.pl1.z + pml.pl2.z    );
  // initCoord(&ran->vr.Vy , x + pml.pl1.x + pml.pl2.x    , y + pml.pl1.y + pml.pl2.y - 1, z + pml.pl1.z + pml.pl2.z    );
  // initCoord(&ran->vr.Vz , x + pml.pl1.x + pml.pl2.x    , y + pml.pl1.y + pml.pl2.y    , z + pml.pl1.z + pml.pl2.z - 1);
  initCoord(&ran->sr.Txx, region.x + pml.pl1.x + pml.pl2.x    , region.y + pml.pl1.y + pml.pl2.y    , region.z + pml.pl1.z + pml.pl2.z    );
  initCoord(&ran->sr.Tyy, region.x + pml.pl1.x + pml.pl2.x    , region.y + pml.pl1.y + pml.pl2.y    , region.z + pml.pl1.z + pml.pl2.z    );
  initCoord(&ran->sr.Tzz, region.x + pml.pl1.x + pml.pl2.x    , region.y + pml.pl1.y + pml.pl2.y    , region.z + pml.pl1.z + pml.pl2.z    );
  initCoord(&ran->tr.Txy, region.x + pml.pl1.x + pml.pl2.x + 1, region.y + pml.pl1.y + pml.pl2.y + 1, region.z + pml.pl1.z + pml.pl2.z    );
  initCoord(&ran->tr.Tyz, region.x + pml.pl1.x + pml.pl2.x    , region.y + pml.pl1.y + pml.pl2.y + 1, region.z + pml.pl1.z + pml.pl2.z + 1);
  initCoord(&ran->tr.Tzx, region.x + pml.pl1.x + pml.pl2.x + 1, region.y + pml.pl1.y + pml.pl2.y    , region.z + pml.pl1.z + pml.pl2.z + 1);
  initCoord(&ran->vr.Vx , region.x + pml.pl1.x + pml.pl2.x + 1, region.y + pml.pl1.y + pml.pl2.y    , region.z + pml.pl1.z + pml.pl2.z    );
  initCoord(&ran->vr.Vy , region.x + pml.pl1.x + pml.pl2.x    , region.y + pml.pl1.y + pml.pl2.y + 1, region.z + pml.pl1.z + pml.pl2.z    );
  initCoord(&ran->vr.Vz , region.x + pml.pl1.x + pml.pl2.x    , region.y + pml.pl1.y + pml.pl2.y    , region.z + pml.pl1.z + pml.pl2.z + 1);
} 

void initrandom(Coord con_size, Coord *clack, int ratio) {
  if(con_size.x < 3 || con_size.y < 3 || con_size.z < 3) {
    printf("Cannot place defects.\n");
    return;
  }
  int count = 0;
  // コンクリートセル数
  int max_Patern = con_size.x * con_size.y * con_size.z;
  // 内部欠陥パターン数
  int max_ClackPatern = (con_size.x - 2) * (con_size.y - 2) * (con_size.z - 2);
  // 割合による欠陥数
  int clack_count = max_Patern * ratio / 100;
  if(clack_count > max_ClackPatern){
    printf("The number of internal defects is insufficient.\n");
    return;
  }
  
  // 乱数の種を初期化
  srand(time(NULL));

  while (count < clack_count) {
    // 新しい乱数の組み合わせを生成
    int rand1 = rand() % (con_size.x - 2) + 1;
    int rand2 = rand() % (con_size.y - 2) + 1;
    int rand3 = rand() % (con_size.z - 2) + 1;

    // 重複がないかチェック
    int is_unique = 1;
    for (int i = 0; i < count; i++) {
      if (clack[i].x == rand1 && clack[i].y == rand2 && clack[i].z == rand3) {
        is_unique = 0;
        break;
      }
    }

    // 重複がなければ保存
    if (is_unique) {
      clack[count].x = rand1;
      clack[count].y = rand2;
      clack[count].z = rand3;
      count++;
    }
  }
}