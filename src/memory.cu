#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "../header/struct.h"
#include "../header/memory.h"


AccCoord* allocateHostAccCoord(int outNum) {
    AccCoord *acccoordptr;
    acccoordptr = (AccCoord*)malloc(outNum * sizeof(AccCoord));

    // acccoordptr->x = (double*)malloc(outNum * sizeof(double));
    // acccoordptr->y = (double*)malloc(outNum * sizeof(double));
    // acccoordptr->z = (double*)malloc(outNum * sizeof(double));

    // printf("%p\n",acccoordptr);
    // printf("%p\n",&acccoordptr->y);
    return acccoordptr;

}
////////////////////////////////////////////////////////////////////////////////////////////
// deviceメモリ確保

BefAft* allocateDeviceBefAft(Range *ran) {
    BefAft *d_befaftptr;// = (BefAft*)malloc(sizeof(BefAft));
    BefAft *h_befaftptr;

    hipMalloc(&d_befaftptr, sizeof(BefAft));
    h_befaftptr = (BefAft*)malloc(sizeof(BefAft));

    // SigArr
    hipMalloc((void**)&(h_befaftptr->sa.Txx ), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->sa.Txxx), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->sa.Txxy), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->sa.Txxz), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));

    hipMalloc((void**)&(h_befaftptr->sa.Tyy ), ran->sr.Tyy.x * ran->sr.Tyy.y * ran->sr.Tyy.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->sa.Tyyx), ran->sr.Tyy.x * ran->sr.Tyy.y * ran->sr.Tyy.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->sa.Tyyy), ran->sr.Tyy.x * ran->sr.Tyy.y * ran->sr.Tyy.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->sa.Tyyz), ran->sr.Tyy.x * ran->sr.Tyy.y * ran->sr.Tyy.z * sizeof(double));

    hipMalloc((void**)&(h_befaftptr->sa.Tzz ), ran->sr.Tzz.x * ran->sr.Tzz.y * ran->sr.Tzz.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->sa.Tzzx), ran->sr.Tzz.x * ran->sr.Tzz.y * ran->sr.Tzz.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->sa.Tzzy), ran->sr.Tzz.x * ran->sr.Tzz.y * ran->sr.Tzz.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->sa.Tzzz), ran->sr.Tzz.x * ran->sr.Tzz.y * ran->sr.Tzz.z * sizeof(double));

    // TauArr
    hipMalloc((void**)&(h_befaftptr->ta.Txy ), ran->tr.Txy.x * ran->tr.Txy.y * ran->tr.Txy.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->ta.Txyx), ran->tr.Txy.x * ran->tr.Txy.y * ran->tr.Txy.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->ta.Txyy), ran->tr.Txy.x * ran->tr.Txy.y * ran->tr.Txy.z * sizeof(double));

    hipMalloc((void**)&(h_befaftptr->ta.Tyz ), ran->tr.Tyz.x * ran->tr.Tyz.y * ran->tr.Tyz.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->ta.Tyzy), ran->tr.Tyz.x * ran->tr.Tyz.y * ran->tr.Tyz.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->ta.Tyzz), ran->tr.Tyz.x * ran->tr.Tyz.y * ran->tr.Tyz.z * sizeof(double));

    hipMalloc((void**)&(h_befaftptr->ta.Tzx ), ran->tr.Tzx.x * ran->tr.Tzx.y * ran->tr.Tzx.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->ta.Tzxz), ran->tr.Tzx.x * ran->tr.Tzx.y * ran->tr.Tzx.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->ta.Tzxx), ran->tr.Tzx.x * ran->tr.Tzx.y * ran->tr.Tzx.z * sizeof(double));

        // VelArr
    hipMalloc((void**)&(h_befaftptr->va.Vx  ), ran->vr.Vx.x * ran->vr.Vx.y * ran->vr.Vx.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->va.Vxx ), ran->vr.Vx.x * ran->vr.Vx.y * ran->vr.Vx.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->va.Vxy ), ran->vr.Vx.x * ran->vr.Vx.y * ran->vr.Vx.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->va.Vxz ), ran->vr.Vx.x * ran->vr.Vx.y * ran->vr.Vx.z * sizeof(double));

    hipMalloc((void**)&(h_befaftptr->va.Vy  ), ran->vr.Vy.x * ran->vr.Vy.y * ran->vr.Vy.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->va.Vyx ), ran->vr.Vy.x * ran->vr.Vy.y * ran->vr.Vy.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->va.Vyy ), ran->vr.Vy.x * ran->vr.Vy.y * ran->vr.Vy.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->va.Vyz ), ran->vr.Vy.x * ran->vr.Vy.y * ran->vr.Vy.z * sizeof(double));

    hipMalloc((void**)&(h_befaftptr->va.Vz  ), ran->vr.Vz.x * ran->vr.Vz.y * ran->vr.Vz.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->va.Vzx ), ran->vr.Vz.x * ran->vr.Vz.y * ran->vr.Vz.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->va.Vzy ), ran->vr.Vz.x * ran->vr.Vz.y * ran->vr.Vz.z * sizeof(double));
    hipMalloc((void**)&(h_befaftptr->va.Vzz ), ran->vr.Vz.x * ran->vr.Vz.y * ran->vr.Vz.z * sizeof(double));
    // Copy SigArr (Stress tensors)
    hipMemcpy(&(d_befaftptr->sa.Txx), &h_befaftptr->sa.Txx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->sa.Txxx), &h_befaftptr->sa.Txxx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->sa.Txxy), &h_befaftptr->sa.Txxy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->sa.Txxz), &h_befaftptr->sa.Txxz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_befaftptr->sa.Tyy), &h_befaftptr->sa.Tyy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->sa.Tyyx), &h_befaftptr->sa.Tyyx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->sa.Tyyy), &h_befaftptr->sa.Tyyy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->sa.Tyyz), &h_befaftptr->sa.Tyyz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_befaftptr->sa.Tzz), &h_befaftptr->sa.Tzz, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->sa.Tzzx), &h_befaftptr->sa.Tzzx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->sa.Tzzy), &h_befaftptr->sa.Tzzy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->sa.Tzzz), &h_befaftptr->sa.Tzzz, sizeof(double*), hipMemcpyHostToDevice);

    // Copy TauArr (Shear stress tensors)
    hipMemcpy(&(d_befaftptr->ta.Txy), &h_befaftptr->ta.Txy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->ta.Txyx), &h_befaftptr->ta.Txyx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->ta.Txyy), &h_befaftptr->ta.Txyy, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_befaftptr->ta.Tyz), &h_befaftptr->ta.Tyz, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->ta.Tyzy), &h_befaftptr->ta.Tyzy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->ta.Tyzz), &h_befaftptr->ta.Tyzz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_befaftptr->ta.Tzx), &h_befaftptr->ta.Tzx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->ta.Tzxz), &h_befaftptr->ta.Tzxz, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->ta.Tzxx), &h_befaftptr->ta.Tzxx, sizeof(double*), hipMemcpyHostToDevice);

    // Copy VelArr (Velocities)
    hipMemcpy(&(d_befaftptr->va.Vx), &h_befaftptr->va.Vx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->va.Vxx), &h_befaftptr->va.Vxx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->va.Vxy), &h_befaftptr->va.Vxy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->va.Vxz), &h_befaftptr->va.Vxz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_befaftptr->va.Vy), &h_befaftptr->va.Vy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->va.Vyx), &h_befaftptr->va.Vyx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->va.Vyy), &h_befaftptr->va.Vyy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->va.Vyz), &h_befaftptr->va.Vyz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&(d_befaftptr->va.Vz), &h_befaftptr->va.Vz, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->va.Vzx), &h_befaftptr->va.Vzx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->va.Vzy), &h_befaftptr->va.Vzy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_befaftptr->va.Vzz), &h_befaftptr->va.Vzz, sizeof(double*), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();
    printf("allocate device befaft: %s\n", hipGetErrorString(err));

    return d_befaftptr;
}

MedArr* allocateDeviceMedArr(Range *ran) {
    MedArr *d_medarrptr;
    MedArr *h_medarrptr;

    hipMalloc(&d_medarrptr, sizeof(MedArr));
    h_medarrptr = (MedArr*)malloc(sizeof(MedArr));

    hipMalloc((void**)&(h_medarrptr->ramda), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->mu), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->c11), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->rho), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));

    hipMalloc((void**)&(h_medarrptr->zetaxx), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->zetaxy), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->zetaxz), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    
    hipMalloc((void**)&(h_medarrptr->zetayx), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->zetayy), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->zetayz), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    
    hipMalloc((void**)&(h_medarrptr->zetazx), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->zetazy), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->zetazz), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));

    hipMalloc((void**)&(h_medarrptr->gamma), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->khi), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->xi11), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));

    hipMalloc((void**)&(h_medarrptr->zetadx), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->zetady), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_medarrptr->zetadz), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));

    hipMemcpy(&d_medarrptr->ramda, &h_medarrptr->ramda, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->mu, &h_medarrptr->mu, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->c11, &h_medarrptr->c11, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->rho, &h_medarrptr->rho, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->zetaxx, &h_medarrptr->zetaxx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetaxy, &h_medarrptr->zetaxy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetaxz, &h_medarrptr->zetaxz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->zetayx, &h_medarrptr->zetayx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetayy, &h_medarrptr->zetayy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetayz, &h_medarrptr->zetayz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->zetazx, &h_medarrptr->zetazx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetazy, &h_medarrptr->zetazy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetazz, &h_medarrptr->zetazz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->gamma, &h_medarrptr->gamma, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->khi, &h_medarrptr->khi, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->xi11, &h_medarrptr->xi11, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->zetadx, &h_medarrptr->zetadx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetady, &h_medarrptr->zetady, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetadz, &h_medarrptr->zetadz, sizeof(double*), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();
    printf("allocate device medarr: %s\n", hipGetErrorString(err));
    // デバイスメモリのポインタを返す
    return d_medarrptr;

}

Impulse* allocateDeviceImpulse(Range *ran) {
    Impulse *d_impulseptr;
    Impulse *h_impulseptr; // ホスト側の一時構造体

    hipMalloc((void**)&d_impulseptr, sizeof(Impulse)); // Impulse構造体のためのメモリをデバイスに確保
    h_impulseptr = (Impulse*)malloc(sizeof(Impulse));

    hipMalloc((void**)&(h_impulseptr->Txx), ran->sr.Txx.x * ran->sr.Txx.y * ran->sr.Txx.z * sizeof(double));
    hipMalloc((void**)&(h_impulseptr->Tyy), ran->sr.Tyy.x * ran->sr.Tyy.y * ran->sr.Tyy.z * sizeof(double));
    hipMalloc((void**)&(h_impulseptr->Tzz), ran->sr.Tzz.x * ran->sr.Tzz.y * ran->sr.Tzz.z * sizeof(double));

    // メモリのポインタをデバイスに戻す
    hipMemcpy(&(d_impulseptr->Txx), &h_impulseptr->Txx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_impulseptr->Tyy), &h_impulseptr->Tyy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(d_impulseptr->Tzz), &h_impulseptr->Tzz, sizeof(double*), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();
    printf("allocate device impulse: %s\n", hipGetErrorString(err));
    return d_impulseptr;
}


AccCoord* allocateDeviceAccCoord(int outNum) {
    // AccCoord *h_acccoordptr;
    AccCoord *d_acccoordptr;
    
    hipMalloc(&d_acccoordptr, outNum * sizeof(AccCoord));  // デバイスメモリにAccCoord配列を確保
    // h_acccoordptr = (AccCoord*)malloc(outNum * sizeof(AccCoord));  // ホストメモリにAccCoord配列を確保

    // // 各AccCoord構造体のx, y, zに対してメモリ確保
    // hipMalloc((void**)&(h_acccoordptr->x), outNum * sizeof(double));
    // hipMalloc((void**)&(h_acccoordptr->y), outNum * sizeof(double));
    // hipMalloc((void**)&(h_acccoordptr->z), outNum * sizeof(double));

    // // ホストからデバイスにx, y, zのポインタをコピー
    // hipMemcpy(&(d_acccoordptr->x), &(h_acccoordptr->x), sizeof(double*), hipMemcpyHostToDevice);
    // hipMemcpy(&(d_acccoordptr->y), &(h_acccoordptr->y), sizeof(double*), hipMemcpyHostToDevice);
    // hipMemcpy(&(d_acccoordptr->z), &(h_acccoordptr->z), sizeof(double*), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();
    printf("allocate device acccoord: %s\n", hipGetErrorString(err));
    return d_acccoordptr;
}


//////////////
// データ転送

void AccCoordDeviceToHost(AccCoord *acc_d, AccCoord *acc_h, int outNum) {
    // AccCoord acc_ptr;
    hipError_t err = hipMemcpy(acc_h, acc_d, outNum * sizeof(double), hipMemcpyDeviceToHost);

    // hipMalloc(&acc_ptr.x, outNum * sizeof(double));
    // hipMalloc(&acc_ptr.y, outNum * sizeof(double));
    // hipMalloc(&acc_ptr.z, outNum * sizeof(double));

    // hipMemcpy(&acc_ptr.x, &acc_d->x, sizeof(double*), hipMemcpyDeviceToHost);
    // hipMemcpy(&acc_ptr.y, &acc_d->y, sizeof(double*), hipMemcpyDeviceToHost);
    // hipMemcpy(&acc_ptr.z, &acc_d->z, sizeof(double*), hipMemcpyDeviceToHost);

    // hipMemcpy(acc_h->x, acc_ptr.x, outNum * sizeof(double), hipMemcpyDeviceToHost);
    // hipMemcpy(acc_h->y, acc_ptr.y, outNum * sizeof(double), hipMemcpyDeviceToHost);
    // err = hipMemcpy(acc_h->z, acc_ptr.z, outNum * sizeof(double), hipMemcpyDeviceToHost);
    // hipError_t err = hipGetLastError();

    printf("acc device to host:%s\n", hipGetErrorString(err));
}

