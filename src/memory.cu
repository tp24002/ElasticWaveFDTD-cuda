#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include "../header/struct.h"
#include "../header/memory.h"

///////////////////////////////
// hostメモリ確保

MedArr* allocateHostMedArr(Range ran) {
    MedArr *medarrptr;
    medarrptr = (MedArr*)malloc(ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(MedArr));
    return medarrptr;
}

// BefAft* allocateHostBefAft(Range *ran) {
//     BefAft *baptr;
//     baptr = (BefAft*)malloc(sizeof(BefAft));
//     // SigArr
//     baptr->sa.Txx = (double*)malloc(ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
//     baptr->sa.Txxx = (double*)malloc(ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
//     baptr->sa.Txxy = (double*)malloc(ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
//     baptr->sa.Txxz = (double*)malloc(ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
//     baptr->sa.Tyy = (double*)malloc(ran.sr.Tyy.x * ran.sr.Tyy.y * ran.sr.Tyy.z * sizeof(double));
//     baptr->sa.Tyyx = (double*)malloc(ran.sr.Tyy.x * ran.sr.Tyy.y * ran.sr.Tyy.z * sizeof(double));
//     baptr->sa.Tyyy = (double*)malloc(ran.sr.Tyy.x * ran.sr.Tyy.y * ran.sr.Tyy.z * sizeof(double));
//     baptr->sa.Tyyz = (double*)malloc(ran.sr.Tyy.x * ran.sr.Tyy.y * ran.sr.Tyy.z * sizeof(double));
//     baptr->sa.Tzz = (double*)malloc(ran.sr.Tzz.x * ran.sr.Tzz.y * ran.sr.Tzz.z * sizeof(double));
//     baptr->sa.Tzzx = (double*)malloc(ran.sr.Tzz.x * ran.sr.Tzz.y * ran.sr.Tzz.z * sizeof(double));
//     baptr->sa.Tzzy = (double*)malloc(ran.sr.Tzz.x * ran.sr.Tzz.y * ran.sr.Tzz.z * sizeof(double));
//     baptr->sa.Tzzz = (double*)malloc(ran.sr.Tzz.x * ran.sr.Tzz.y * ran.sr.Tzz.z * sizeof(double));
//     // TauArr
//     baptr->ta.Txy = (double*)malloc(ran.tr.Txy.x * ran.tr.Txy.y * ran.tr.Txy.z * sizeof(double));
//     baptr->ta.Txyx = (double*)malloc(ran.tr.Txy.x * ran.tr.Txy.y * ran.tr.Txy.z * sizeof(double));
//     baptr->ta.Txyy = (double*)malloc(ran.tr.Txy.x * ran.tr.Txy.y * ran.tr.Txy.z * sizeof(double));
//     baptr->ta.Tyz = (double*)malloc(ran.tr.Tyz.x * ran.tr.Tyz.y * ran.tr.Tyz.z * sizeof(double));
//     baptr->ta.Tyzy = (double*)malloc(ran.tr.Tyz.x * ran.tr.Tyz.y * ran.tr.Tyz.z * sizeof(double));
//     baptr->ta.Tyzz = (double*)malloc(ran.tr.Tyz.x * ran.tr.Tyz.y * ran.tr.Tyz.z * sizeof(double));
//     baptr->ta.Tzx = (double*)malloc(ran.tr.Tzx.x * ran.tr.Tzx.y * ran.tr.Tzx.z * sizeof(double));
//     baptr->ta.Tzxz = (double*)malloc(ran.tr.Tzx.x * ran.tr.Tzx.y * ran.tr.Tzx.z * sizeof(double));
//     baptr->ta.Tzxx = (double*)malloc(ran.tr.Tzx.x * ran.tr.Tzx.y * ran.tr.Tzx.z * sizeof(double));
//     // VelArr
//     baptr->va.Vx = (double*)malloc(ran.vr.Vx.x * ran.vr.Vx.y * ran.vr.Vx.z * sizeof(double));
//     baptr->va.Vxx = (double*)malloc(ran.vr.Vx.x * ran.vr.Vx.y * ran.vr.Vx.z * sizeof(double));
//     baptr->va.Vxy = (double*)malloc(ran.vr.Vx.x * ran.vr.Vx.y * ran.vr.Vx.z * sizeof(double));
//     baptr->va.Vxz = (double*)malloc(ran.vr.Vx.x * ran.vr.Vx.y * ran.vr.Vx.z * sizeof(double));
//     baptr->va.Vy = (double*)malloc(ran.vr.Vy.x * ran.vr.Vy.y * ran.vr.Vy.z * sizeof(double));
//     baptr->va.Vyx = (double*)malloc(ran.vr.Vy.x * ran.vr.Vy.y * ran.vr.Vy.z * sizeof(double));
//     baptr->va.Vyy = (double*)malloc(ran.vr.Vy.x * ran.vr.Vy.y * ran.vr.Vy.z * sizeof(double));
//     baptr->va.Vyz = (double*)malloc(ran.vr.Vy.x * ran.vr.Vy.y * ran.vr.Vy.z * sizeof(double));
//     baptr->va.Vz = (double*)malloc(ran.vr.Vz.x * ran.vr.Vz.y * ran.vr.Vz.z * sizeof(double));
//     baptr->va.Vzx = (double*)malloc(ran.vr.Vz.x * ran.vr.Vz.y * ran.vr.Vz.z * sizeof(double));
//     baptr->va.Vzy = (double*)malloc(ran.vr.Vz.x * ran.vr.Vz.y * ran.vr.Vz.z * sizeof(double));
//     baptr->va.Vzz = (double*)malloc(ran.vr.Vz.x * ran.vr.Vz.y * ran.vr.Vz.z * sizeof(double));
//     return baptr;
// }

Impulse* allocateHostImpulse(int innum) {
    Impulse *impulseptr;
    impulseptr = (Impulse*)malloc(innum * sizeof(Impulse));
    // impulseptr->Txx = (double*)malloc(ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
    // impulseptr->Tyy = (double*)malloc(ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
    // impulseptr->Tzz = (double*)malloc(ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
    // impulseptr->in  = (DimI3*)malloc(innum * sizeof(DimI3));
    return impulseptr;
}

DimI3* allocateHostDimI3(int outnum) {
    DimI3 *DI;
    DI = (DimI3*)malloc(outnum * sizeof(DimI3));
    return DI;
}

DimD3* allocateHostDimD3(int outnum) {
    DimD3 *DD;
    DD = (DimD3*)malloc(outnum * sizeof(DimD3));
    return DD;
}

////////////////////////////////////////////////////////////////////////////////////////////
// deviceメモリ確保
MedArr* allocateDeviceMedArr(Range ran) {
    MedArr *med;
    hipError_t err = hipMalloc(&med, ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(MedArr));
    printf("allocateDeviceMedArr:%s\n", hipGetErrorString(err));
    return med;
}

BefAft* allocateDeviceBefAft(Range ran) {
    BefAft *ba;
    
    ba = (BefAft*)malloc(sizeof(BefAft));

    // SigArr
    hipError_t err = hipMalloc(&ba->sa.Txx , ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
    printf("allocateDeviceBefAft:%s\n", hipGetErrorString(err));
    hipMalloc(&ba->sa.Txxx, ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
    hipMalloc(&ba->sa.Txxy, ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
    hipMalloc(&ba->sa.Txxz, ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));

    hipMalloc(&ba->sa.Tyy , ran.sr.Tyy.x * ran.sr.Tyy.y * ran.sr.Tyy.z * sizeof(double));
    hipMalloc(&ba->sa.Tyyx, ran.sr.Tyy.x * ran.sr.Tyy.y * ran.sr.Tyy.z * sizeof(double));
    hipMalloc(&ba->sa.Tyyy, ran.sr.Tyy.x * ran.sr.Tyy.y * ran.sr.Tyy.z * sizeof(double));
    hipMalloc(&ba->sa.Tyyz, ran.sr.Tyy.x * ran.sr.Tyy.y * ran.sr.Tyy.z * sizeof(double));

    hipMalloc(&ba->sa.Tzz , ran.sr.Tzz.x * ran.sr.Tzz.y * ran.sr.Tzz.z * sizeof(double));
    hipMalloc(&ba->sa.Tzzx, ran.sr.Tzz.x * ran.sr.Tzz.y * ran.sr.Tzz.z * sizeof(double));
    hipMalloc(&ba->sa.Tzzy, ran.sr.Tzz.x * ran.sr.Tzz.y * ran.sr.Tzz.z * sizeof(double));
    hipMalloc(&ba->sa.Tzzz, ran.sr.Tzz.x * ran.sr.Tzz.y * ran.sr.Tzz.z * sizeof(double));

    // TauArr
    hipMalloc(&ba->ta.Txy , ran.tr.Txy.x * ran.tr.Txy.y * ran.tr.Txy.z * sizeof(double));
    hipMalloc(&ba->ta.Txyx, ran.tr.Txy.x * ran.tr.Txy.y * ran.tr.Txy.z * sizeof(double));
    hipMalloc(&ba->ta.Txyy, ran.tr.Txy.x * ran.tr.Txy.y * ran.tr.Txy.z * sizeof(double));

    hipMalloc(&ba->ta.Tyz , ran.tr.Tyz.x * ran.tr.Tyz.y * ran.tr.Tyz.z * sizeof(double));
    hipMalloc(&ba->ta.Tyzy, ran.tr.Tyz.x * ran.tr.Tyz.y * ran.tr.Tyz.z * sizeof(double));
    hipMalloc(&ba->ta.Tyzz, ran.tr.Tyz.x * ran.tr.Tyz.y * ran.tr.Tyz.z * sizeof(double));

    hipMalloc(&ba->ta.Tzx , ran.tr.Tzx.x * ran.tr.Tzx.y * ran.tr.Tzx.z * sizeof(double));
    hipMalloc(&ba->ta.Tzxz, ran.tr.Tzx.x * ran.tr.Tzx.y * ran.tr.Tzx.z * sizeof(double));
    hipMalloc(&ba->ta.Tzxx, ran.tr.Tzx.x * ran.tr.Tzx.y * ran.tr.Tzx.z * sizeof(double));

     // VelArr
    hipMalloc(&ba->va.Vx  , ran.vr.Vx.x * ran.vr.Vx.y * ran.vr.Vx.z * sizeof(double));
    hipMalloc(&ba->va.Vxx , ran.vr.Vx.x * ran.vr.Vx.y * ran.vr.Vx.z * sizeof(double));
    hipMalloc(&ba->va.Vxy , ran.vr.Vx.x * ran.vr.Vx.y * ran.vr.Vx.z * sizeof(double));
    hipMalloc(&ba->va.Vxz , ran.vr.Vx.x * ran.vr.Vx.y * ran.vr.Vx.z * sizeof(double));

    hipMalloc(&ba->va.Vy  , ran.vr.Vy.x * ran.vr.Vy.y * ran.vr.Vy.z * sizeof(double));
    hipMalloc(&ba->va.Vyx , ran.vr.Vy.x * ran.vr.Vy.y * ran.vr.Vy.z * sizeof(double));
    hipMalloc(&ba->va.Vyy , ran.vr.Vy.x * ran.vr.Vy.y * ran.vr.Vy.z * sizeof(double));
    hipMalloc(&ba->va.Vyz , ran.vr.Vy.x * ran.vr.Vy.y * ran.vr.Vy.z * sizeof(double));

    hipMalloc(&ba->va.Vz  , ran.vr.Vz.x * ran.vr.Vz.y * ran.vr.Vz.z * sizeof(double));
    hipMalloc(&ba->va.Vzx , ran.vr.Vz.x * ran.vr.Vz.y * ran.vr.Vz.z * sizeof(double));
    hipMalloc(&ba->va.Vzy , ran.vr.Vz.x * ran.vr.Vz.y * ran.vr.Vz.z * sizeof(double));
    hipMalloc(&ba->va.Vzz , ran.vr.Vz.x * ran.vr.Vz.y * ran.vr.Vz.z * sizeof(double));

    return ba;
}

// ImpulseArr* allocateDeviceImpulseArr(Range ran) {
//     ImpulseArr *ip;
//     ip = (ImpulseArr*)malloc(sizeof(ImpulseArr));
//     hipError_t err = hipMalloc(&ip->Txx, ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(double));
//     printf("allocateDeviceImpulseArr:%s\n", hipGetErrorString(err));
//     hipMalloc(&ip->Tyy, ran.sr.Tyy.x * ran.sr.Tyy.y * ran.sr.Tyy.z * sizeof(double));
//     hipMalloc(&ip->Tzz, ran.sr.Tzz.x * ran.sr.Tzz.y * ran.sr.Tzz.z * sizeof(double));
//     return ip;
// }

ImpulseArr* allocateDeviceImpulseArr(Range ran) {
    ImpulseArr *ipa;
    ipa = (ImpulseArr*)malloc(sizeof(ImpulseArr));
    hipError_t err = hipMalloc(&ipa, ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(ImpulseArr));
    printf("allocateDeviceImpulseArr:%s\n", hipGetErrorString(err));
    return ipa;
}

Impulse* allocateDeviceImpulse(int innum) {
    Impulse *ip; // ホスト側の一時構造体
    hipError_t err = hipMalloc(&ip, innum * sizeof(Impulse));
    printf("allocateDeviceImpulse:%s\n", hipGetErrorString(err));
    return ip;
}

DimI3* allocateDeviceDimI3(int outNum) {
    DimI3 *DI;
    hipError_t err = hipMalloc(&DI, outNum * sizeof(DimD3));
    printf("allocateDeviceDimI3:%s\n", hipGetErrorString(err));
    return DI;
}

DimD3* allocateDeviceDimD3(int outNum) {
    DimD3 *DD;
    hipError_t err = hipMalloc(&DD, outNum * sizeof(DimD3));
    printf("allocateDeviceDimD3:%s\n", hipGetErrorString(err));
    return DD;
}


//////////////
// データ転送
// host to device
void RangeHostToDevice(Range *ran_d, Range *ran_h) {
    hipMemcpy(ran_d, ran_h, sizeof(Range), hipMemcpyHostToDevice);
}

void DiffHostToDevice(Diff *dif_d, Diff *dif_h) {
    hipMemcpy(dif_d, dif_h, sizeof(Diff), hipMemcpyHostToDevice);
}

void MedArrHostToDevice(MedArr *ma_d, MedArr *ma_h, Range ran) {
    hipMemcpy(ma_d, ma_h, ran.sr.Txx.x * ran.sr.Txx.y * ran.sr.Txx.z * sizeof(MedArr), hipMemcpyHostToDevice);
}

void ImpulseHostToDevice(Impulse *ip_d, Impulse *ip_h, int innum) {
    hipMemcpy(ip_d, ip_h, innum * sizeof(Impulse), hipMemcpyHostToDevice);
}

void DimI3HostToDevice(DimI3 *di_d, DimI3 *di_h, int outnum) {
    hipMemcpy(di_d, di_h, outnum * sizeof(DimI3), hipMemcpyHostToDevice);
}

// device to host

void DimD3DeviceToHost(DimD3 *acc_h, DimD3 *acc_d, int outNum) {
    hipMemcpy(acc_h, acc_d, outNum * sizeof(DimD3), hipMemcpyDeviceToHost);
}

