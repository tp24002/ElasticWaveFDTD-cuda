#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include "../header/update.h"
#include "../header/struct.h"

#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

#include "../header/init.h"
#include "../header/memory.h"

// 垂直応力

// 垂直応力更新並列関数
__global__ void TxxUpdate(BefAft *aft, BefAft *bef, MedArr *ma, Diff *dif, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

  int imax = ran->sr.Txx.x, jmax = ran->sr.Txx.y, kmax = ran->sr.Txx.z;

  if(i < imax - 1 && j < jmax - 1 && k < kmax - 1) {
    int idx    = k * imax * jmax + j * imax + i;
    int idx_i1 = k * imax * jmax + j * imax + (i + 1);
    int idx_j1 = k * imax * jmax + (j + 1) * imax + i;
    int idx_k1 = (k + 1) * imax * jmax + j * imax + i;

    aft->sa.Txxx[idx] = (2.0 - ma->zetadx[idx] * dif->dt) / (2.0 + ma->zetadx[idx] * dif->dt) * bef->sa.Txxx[idx]
        + 2.0 * (ma->c11[idx] * dif->dt + ma->xi11[idx]) / (2.0 + ma->zetadx[idx] * dif->dt) * (aft->va.Vx[idx_i1] - aft->va.Vx[idx]) / dif->dx
        - 2.0 * ma->xi11[idx] / (2.0 + ma->zetadx[idx] * dif->dt) * (bef->va.Vx[idx_i1] - bef->va.Vx[idx]) / dif->dx;

    aft->sa.Txxy[idx] = (2.0 - ma->zetady[idx] * dif->dt) / (2.0 + ma->zetady[idx] * dif->dt) * bef->sa.Txxy[idx]
        + 2.0 * (ma->ramda[idx] * dif->dt + ma->khi[idx]) / (2.0 + ma->zetady[idx] * dif->dt) * (aft->va.Vy[idx_j1] - aft->va.Vy[idx]) / dif->dy
        - 2.0 * ma->khi[idx] / (2.0 + ma->zetady[idx] * dif->dt) * (bef->va.Vy[idx_j1] - bef->va.Vy[idx]) / dif->dy;

    aft->sa.Txxz[idx] = (2.0 - ma->zetadz[idx] * dif->dt) / (2.0 + ma->zetadz[idx] * dif->dt) * bef->sa.Txxz[idx]
        + 2.0 * (ma->ramda[idx] * dif->dt + ma->khi[idx]) / (2.0 + ma->zetadz[idx] * dif->dt) * (aft->va.Vz[idx_k1] - aft->va.Vz[idx]) / dif->dz
        - 2.0 * ma->khi[idx] / (2.0 + ma->zetadz[idx] * dif->dt) * (bef->va.Vz[idx_k1] - bef->va.Vz[idx]) / dif->dz;
  }
}
// 垂直応力更新並列関数
__global__ void TyyUpdate(BefAft *aft, BefAft *bef, MedArr *ma, Diff *dif, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

  int imax = ran->sr.Tyy.x, jmax = ran->sr.Tyy.y, kmax = ran->sr.Tyy.z;

  if(i < imax - 1 && j < jmax - 1 && k < kmax - 1) {
    int idx    = k * imax * jmax + j * imax + i;
    int idx_i1 = k * imax * jmax + j * imax + (i + 1);
    int idx_j1 = k * imax * jmax + (j + 1) * imax + i;
    int idx_k1 = (k + 1) * imax * jmax + j * imax + i;

    aft->sa.Tyyx[idx] = (2.0 - ma->zetadx[idx] * dif->dt) / (2.0 + ma->zetadx[idx] * dif->dt) * bef->sa.Tyyx[idx]
        + 2.0 * (ma->ramda[idx] * dif->dt + ma->khi[idx]) / (2.0 + ma->zetadx[idx] * dif->dt) * (aft->va.Vx[idx_i1] - aft->va.Vx[idx]) / dif->dx
        - 2.0 * ma->khi[idx] / (2.0 + ma->zetadx[idx] * dif->dt) * (bef->va.Vx[idx_i1] - bef->va.Vx[idx]) / dif->dx;

    aft->sa.Tyyy[idx] = (2.0 - ma->zetady[idx] * dif->dt) / (2.0 + ma->zetady[idx] * dif->dt) * bef->sa.Tyyy[idx]
        + 2.0 * (ma->c11[idx] * dif->dt + ma->xi11[idx]) / (2.0 + ma->zetady[idx] * dif->dt) * (aft->va.Vy[idx_j1] - aft->va.Vy[idx]) / dif->dy
        - 2.0 * ma->xi11[idx] / (2.0 + ma->zetady[idx] * dif->dt) * (bef->va.Vy[idx_j1] - bef->va.Vy[idx]) / dif->dy;

    aft->sa.Tyyz[idx] = (2.0 - ma->zetadz[idx] * dif->dt) / (2.0 + ma->zetadz[idx] * dif->dt) * bef->sa.Tyyz[idx]
        + 2.0 * (ma->ramda[idx] * dif->dt + ma->khi[idx]) / (2.0 + ma->zetadz[idx] * dif->dt) * (aft->va.Vz[idx_k1] - aft->va.Vz[idx]) / dif->dz
        - 2.0 * ma->khi[idx] / (2.0 + ma->zetadz[idx] * dif->dt) * (bef->va.Vz[idx_k1] - bef->va.Vz[idx]) / dif->dz;
  }
}
// 垂直応力更新並列関数
__global__ void TzzUpdate(BefAft *aft, BefAft *bef, MedArr *ma, Diff *dif, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

  int imax = ran->sr.Tzz.x, jmax = ran->sr.Tzz.y, kmax = ran->sr.Tzz.z;
  
  if(i < imax - 1 && j < jmax - 1 && k < kmax - 1) {
    int idx     = k * imax * jmax + j * imax + i;
    int idx_im1 = k * imax * jmax + j * imax + (i + 1);
    int idx_jm1 = k * imax * jmax + (j + 1) * imax + i;
    int idx_km1 = (k + 1) * imax * jmax + j * imax + i;

    aft->sa.Tzzx[idx] = (2.0 - ma->zetadx[idx] * dif->dt) / (2.0 + ma->zetadx[idx] * dif->dt) * bef->sa.Tzzx[idx]
        + 2.0 * (ma->ramda[idx] * dif->dt + ma->khi[idx]) / (2.0 + ma->zetadx[idx] * dif->dt) * (aft->va.Vx[idx_im1] - aft->va.Vx[idx]) / dif->dx
        - 2.0 * ma->khi[idx] / (2.0 + ma->zetadx[idx] * dif->dt) * (bef->va.Vx[idx_im1] - bef->va.Vx[idx]) / dif->dx;

    aft->sa.Tzzy[idx] = (2.0 - ma->zetady[idx] * dif->dt) / (2.0 + ma->zetady[idx] * dif->dt) * bef->sa.Tzzy[idx]
        + 2.0 * (ma->ramda[idx] * dif->dt + ma->khi[idx]) / (2.0 + ma->zetady[idx] * dif->dt) * (aft->va.Vy[idx_jm1] - aft->va.Vy[idx]) / dif->dy
        - 2.0 * ma->khi[idx] / (2.0 + ma->zetady[idx] * dif->dt) * (bef->va.Vy[idx_jm1] - bef->va.Vy[idx]) / dif->dy;

    aft->sa.Tzzz[idx] = (2.0 - ma->zetadz[idx] * dif->dt) / (2.0 + ma->zetadz[idx] * dif->dt) * bef->sa.Tzzz[idx]
        + 2.0 * (ma->c11[idx] * dif->dt + ma->xi11[idx]) / (2.0 + ma->zetadz[idx] * dif->dt) * (aft->va.Vz[idx_km1] - aft->va.Vz[idx]) / dif->dz
        - 2.0 * ma->xi11[idx] / (2.0 + ma->zetadz[idx] * dif->dt) * (bef->va.Vz[idx_km1] - bef->va.Vz[idx]) / dif->dz;
  }
}
// 全方向加算
__global__ void DirectionalAdd(BefAft *aft, Impulse *ip, Range *ran, char check) {
  // 1Dインデックスの計算
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  int imax = ran->sr.Txx.x, jmax = ran->sr.Txx.y, kmax = ran->sr.Txx.z;

  // 1Dインデックス化
  int idx = k * imax * jmax + j * imax + i;
  // printf("%lf\n",*ip->Txx);
  if (i < imax && j < jmax && k < kmax) {
    // 各方向に応じた計算を実行（ポインタ表記）
    if (check == 'X') {
      aft->sa.Txx[idx] = aft->sa.Txxx[idx] + aft->sa.Txxy[idx] + aft->sa.Txxz[idx] + ip->Txx[idx];
    } else if (check == 'Y') {
      aft->sa.Tyy[idx] = aft->sa.Tyyx[idx] + aft->sa.Tyyy[idx] + aft->sa.Tyyz[idx] + ip->Tyy[idx];
    } else if (check == 'Z') {
      aft->sa.Tzz[idx] = aft->sa.Tzzx[idx] + aft->sa.Tzzy[idx] + aft->sa.Tzzz[idx] + ip->Tzz[idx];
    } else {
      printf("error: DirectionalAdd\n");
    }
  }
}

// Txxクラス的な(Blocks大丈夫かな？)
void Txx(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Impulse *ip_d, Coord threads) {
  // hipError_t err;
  // char check = 'X';

  int Txximax = ran_h->sr.Txx.x, Txxjmax = ran_h->sr.Txx.y, Txxkmax = ran_h->sr.Txx.z;

  dim3 threadsPerBlock(threads.x, threads.y, threads.z);  // ブロック内のスレッド数
  dim3 UpdateBlocks((Txximax - 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (Txxjmax - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (Txxkmax - 2 + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // dim3 ZeroXYBlocks((Txximax     + threadsPerBlock.x - 1) / threadsPerBlock.x, (Txxjmax     + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // dim3 ZeroYZBlocks((Txxjmax     + threadsPerBlock.x - 1) / threadsPerBlock.x, (Txxkmax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // dim3 ZeroZXBlocks((Txxkmax - 1 + threadsPerBlock.x - 1) / threadsPerBlock.x, (Txximax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
  dim3 DirectionalAddBlocks((Txximax + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (Txxjmax + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                            (Txxkmax + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // Txx更新式
  TxxUpdate<<<UpdateBlocks, threadsPerBlock>>>(aft_d, bef_d, ma_d, dif_d, ran_d);
  // 0 padding
  // ZeroT_XY<<<ZeroXYBlocks, threadsPerBlock>>>(aft_d, ran_d, 'X');
  // ZeroT_YZ<<<ZeroYZBlocks, threadsPerBlock>>>(aft_d, ran_d, 'X');
  // ZeroT_ZX<<<ZeroZXBlocks, threadsPerBlock>>>(aft_d, ran_d, 'X');
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Txx zero  : %s\n", hipGetErrorString(err));
  //全方向加算
  DirectionalAdd<<<DirectionalAddBlocks, threadsPerBlock>>>(aft_d, ip_d, ran_d, 'X');
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Txx add   : %s\n", hipGetErrorString(err));

}
// Tyyクラス的な(Blocks大丈夫かな？)
void Tyy(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Impulse *ip_d, Coord threads) {
  // hipError_t err;
  char check = 'Y';

  int Tyyimax = ran_h->sr.Tyy.x, Tyyjmax = ran_h->sr.Tyy.y, Tyykmax = ran_h->sr.Tyy.z;

  dim3 threadsPerBlock(threads.x, threads.y, threads.z);  // ブロック内のスレッド数
  dim3 UpdateBlocks((Tyyimax - 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (Tyyjmax - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (Tyykmax - 2 + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // dim3 ZeroXYBlocks((Tyyimax     + threadsPerBlock.x - 1) / threadsPerBlock.x, (Tyyjmax     + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // dim3 ZeroYZBlocks((Tyyjmax     + threadsPerBlock.x - 1) / threadsPerBlock.x, (Tyykmax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // dim3 ZeroZXBlocks((Tyykmax - 1 + threadsPerBlock.x - 1) / threadsPerBlock.x, (Tyyimax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
  dim3 DirectionalAddBlocks((Tyyimax + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (Tyyjmax + threadsPerBlock.y - 1) / threadsPerBlock.y,
                            (Tyykmax + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // Tyy更新式
  TyyUpdate<<<UpdateBlocks, threadsPerBlock>>>(aft_d, bef_d, ma_d, dif_d, ran_d);
  // 0 padding
  // ZeroT_XY<<<ZeroXYBlocks, threadsPerBlock>>>(aft_d, ran_d, check);
  // ZeroT_YZ<<<ZeroYZBlocks, threadsPerBlock>>>(aft_d, ran_d, check);
  // ZeroT_ZX<<<ZeroZXBlocks, threadsPerBlock>>>(aft_d, ran_d, check);
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tyy zero  : %s\n", hipGetErrorString(err));
  // 全方向加算
  DirectionalAdd<<<DirectionalAddBlocks, threadsPerBlock>>>(aft_d, ip_d, ran_d, check);
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tyy add   : %s\n", hipGetErrorString(err));
}
// Tzzクラス的な(Blocks大丈夫かな？)
void Tzz(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Impulse *ip_d, Coord threads) {
  // hipError_t err;
  char check = 'Z';

  int Tzzimax = ran_h->sr.Tzz.x, Tzzjmax = ran_h->sr.Tzz.y, Tzzkmax = ran_h->sr.Tzz.z;

  dim3 threadsPerBlock(threads.x, threads.y, threads.z);  // ブロック内のスレッド数
  dim3 UpdateBlocks((Tzzimax - 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (Tzzjmax - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (Tzzkmax - 2 + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // dim3 ZeroXYBlocks((Tzzimax     + threadsPerBlock.x - 1) / threadsPerBlock.x, (Tzzjmax     + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // dim3 ZeroYZBlocks((Tzzjmax     + threadsPerBlock.x - 1) / threadsPerBlock.x, (Tzzkmax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // dim3 ZeroZXBlocks((Tzzkmax - 1 + threadsPerBlock.x - 1) / threadsPerBlock.x, (Tzzimax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
  dim3 DirectionalAddBlocks((Tzzimax + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (Tzzjmax + threadsPerBlock.y - 1) / threadsPerBlock.y,
                            (Tzzkmax + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // Tzzの更新式
  TzzUpdate<<<UpdateBlocks, threadsPerBlock>>>(aft_d, bef_d, ma_d, dif_d, ran_d);
  // 0 padding
  // ZeroT_XY<<<ZeroXYBlocks, threadsPerBlock>>>(aft_d, ran_d, check);
  // ZeroT_YZ<<<ZeroYZBlocks, threadsPerBlock>>>(aft_d, ran_d, check);
  // ZeroT_ZX<<<ZeroZXBlocks, threadsPerBlock>>>(aft_d, ran_d, check);
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tzz zero  : %s\n", hipGetErrorString(err));
  // 全方向加算
  DirectionalAdd<<<DirectionalAddBlocks, threadsPerBlock>>>(aft_d, ip_d, ran_d, check);
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tzz add   : %s\n", hipGetErrorString(err));
 
}
// 垂直応力計算(main呼び出し関数)
void Sig(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Impulse *ip_d, Coord threads) {
  Txx(aft_d, bef_d, ma_d, dif_d, ran_d, ran_h, ip_d, threads);
  Tyy(aft_d, bef_d, ma_d, dif_d, ran_d, ran_h, ip_d, threads);
  Tzz(aft_d, bef_d, ma_d, dif_d, ran_d, ran_h, ip_d, threads);
}

// せん断応力

// せん断応力更新関数
__global__ void TxyUpdate(BefAft *aft, BefAft *bef, MedArr *ma, Diff *dif, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1; // 始点を+1

  int imax = ran->tr.Txy.x, jmax = ran->tr.Txy.y, kmax = ran->tr.Txy.z;
  double Hzetadx, Hzetady, Hmu, Hgamma;

  if (i < imax - 1 && j < jmax - 1 && k < kmax - 1) {
    // 各インデックスの計算
    int idx     = k * imax * jmax + j * imax + i;
    int idx_i1  = k * imax * jmax + j * imax + (i - 1);
    int idx_j1  = k * imax * jmax + (j - 1) * imax + i;
    int idx_ij1 = k * imax * jmax + (j - 1) * imax + (i - 1);

    // PML:減衰係数,計算領域:摩擦定数
    Hzetadx = 4. * pow((1. / ma->zetadx[idx_i1]) + (1. / ma->zetadx[idx_j1]) + (1. / ma->zetadx[idx_ij1]) + (1. / ma->zetadx[idx]), -1.);
    Hzetady = 4. * pow((1. / ma->zetady[idx_i1]) + (1. / ma->zetady[idx_j1]) + (1. / ma->zetady[idx_ij1]) + (1. / ma->zetady[idx]), -1.);
    // 第2ラメ，横弾性係数(剛性率)
    Hmu = 4. * pow((1. / ma->mu[idx_i1]) + (1. / ma->mu[idx_j1]) + (1. / ma->mu[idx_ij1]) + (1. / ma->mu[idx]), -1.);
    // 第1粘性定数
    Hgamma = 4. * pow((1. / ma->gamma[idx_i1]) + (1. / ma->gamma[idx_j1]) + (1. / ma->gamma[idx_ij1]) + (1. / ma->gamma[idx]), -1.);


    aft->ta.Txyx[idx] = (2.0 - Hzetadx * dif->dt) / (2.0 + Hzetadx * dif->dt) * bef->ta.Txyx[idx]
        + 2.0 * (Hmu * dif->dt + Hgamma) / (2.0 + Hzetadx * dif->dt) * (aft->va.Vy[idx] - aft->va.Vy[idx_i1]) / dif->dx
        - 2.0 * Hgamma / (2.0 + Hzetadx * dif->dt) * (bef->va.Vy[idx] - bef->va.Vy[idx_i1]) / dif->dx;

    aft->ta.Txyy[idx] = (2.0 - Hzetady * dif->dt) / (2.0 + Hzetady * dif->dt) * bef->ta.Txyy[idx]
        + 2.0 * (Hmu * dif->dt + Hgamma) / (2.0 + Hzetady * dif->dt) * (aft->va.Vx[idx] - aft->va.Vx[idx_j1]) / dif->dy
        - 2.0 * Hgamma / (2.0 + Hzetady * dif->dt) * (bef->va.Vx[idx] - bef->va.Vx[idx_j1]) / dif->dy;
  }
}
// せん断応力更新関数
__global__ void TyzUpdate(BefAft *aft, BefAft *bef, MedArr *ma, Diff *dif, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

  int imax = ran->tr.Tyz.x, jmax = ran->tr.Tyz.y, kmax = ran->tr.Tyz.z;
  double Hzetady, Hzetadz, Hmu, Hgamma;

  if (i < imax - 1 && j < jmax - 1 && k < kmax - 1) {
    // 各インデックスの計算
    int idx      = k * imax * jmax + j * imax + i;
    int idx_j1  = k * imax * jmax + (j - 1) * imax + i;
    int idx_k1  = (k - 1) * imax * jmax + j * imax + i;
    int idx_jk1 = (k - 1) * jmax * kmax + (j - 1) * kmax + i;

    // PML:減衰係数,計算領域:摩擦定数
    Hzetady = 4. * pow((1. / ma->zetady[idx_jk1]) + (1. / ma->zetady[idx_j1]) + (1. / ma->zetady[idx_k1]) + (1. / ma->zetady[idx]), -1.);
    Hzetadz = 4. * pow((1. / ma->zetadz[idx_jk1]) + (1. / ma->zetadz[idx_j1]) + (1. / ma->zetadz[idx_k1]) + (1. / ma->zetadz[idx]), -1.);
    // 第2ラメ，横弾性係数(剛性率)
    Hmu = 4. * pow((1. / ma->mu[idx_jk1]) + (1. / ma->mu[idx_j1]) + (1. / ma->mu[idx_k1]) + (1. / ma->mu[idx]), -1.);
    // 第1粘性定数
    Hgamma = 4. * pow((1. / ma->gamma[idx_jk1]) + (1. / ma->gamma[idx_j1]) + (1. / ma->gamma[idx_k1]) + (1. / ma->gamma[idx]), -1.);

    aft->ta.Tyzy[idx] = (2.0 - Hzetady * dif->dt) / (2.0 + Hzetady * dif->dt) * bef->ta.Tyzy[idx]
        + 2.0 * (Hmu * dif->dt + Hgamma) / (2.0 + Hzetady * dif->dt) * (aft->va.Vz[idx] - aft->va.Vz[idx_j1]) / dif->dy
        - 2.0 * Hgamma / (2.0 + Hzetady * dif->dt) * (bef->va.Vz[idx] - bef->va.Vz[idx_j1]) / dif->dy;

    aft->ta.Tyzz[idx] = (2.0 - Hzetadz * dif->dt) / (2.0 + Hzetadz * dif->dt) * bef->ta.Tyzz[idx]
        + 2.0 * (Hmu * dif->dt + Hgamma) / (2.0 + Hzetadz * dif->dt) * (aft->va.Vy[idx] - aft->va.Vy[idx_k1]) / dif->dz
        - 2.0 * Hgamma / (2.0 + Hzetadz * dif->dt) * (bef->va.Vy[idx] - bef->va.Vy[idx_k1]) / dif->dz;

  }
}
// せん断応力更新関数
__global__ void TzxUpdate(BefAft *aft, BefAft *bef, MedArr *ma, Diff *dif, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

  int imax = ran->tr.Tzx.x, jmax = ran->tr.Tzx.y, kmax = ran->tr.Tzx.z;
  double Hzetadx, Hzetadz, Hmu, Hgamma;

  if (i < imax - 1 && j < jmax - 1 && k < kmax - 1) {
    // 各インデックスの計算
    int idx      = k * imax * jmax + j * imax + i;
    int idx_i1  = k * imax * jmax + j * imax + (i - 1);
    int idx_k1  = (k - 1) * imax * jmax + j * imax + i;
    int idx_ki1 = (k - 1) * imax * imax + j * imax + (i - 1);

    // PML:減衰係数,計算領域:摩擦定数
    Hzetadx = 4. * pow((1. / ma->zetadx[idx_ki1]) + (1. / ma->zetadx[idx_i1]) + (1. / ma->zetadx[idx_k1]) + (1. / ma->zetadx[idx]), -1.);
    Hzetadz = 4. * pow((1. / ma->zetadz[idx_ki1]) + (1. / ma->zetadz[idx_i1]) + (1. / ma->zetadz[idx_k1]) + (1. / ma->zetadz[idx]), -1.);
    // 第2ラメ，横弾性係数(剛性率)
    Hmu = 4. * pow((1. / ma->mu[idx_ki1]) + (1. / ma->mu[idx_i1]) + (1. / ma->mu[idx_k1]) + (1. / ma->mu[idx]), -1.);
    // 第1粘性定数
    Hgamma = 4. * pow((1. / ma->gamma[idx_ki1]) + (1. / ma->gamma[idx_i1]) + (1. / ma->gamma[idx_k1]) + (1. / ma->gamma[idx]), -1.);


    aft->ta.Tzxz[idx] = (2.0 - Hzetadz * dif->dt) / (2.0 + Hzetadz * dif->dt) * bef->ta.Tzxz[idx]
        + 2.0 * (Hmu * dif->dt + Hgamma) / (2.0 + Hzetadz * dif->dt) * (aft->va.Vx[idx] - aft->va.Vx[idx_k1]) / dif->dz
        - 2.0 * Hgamma / (2.0 + Hzetadz * dif->dt) * (bef->va.Vx[idx] - bef->va.Vx[idx_k1]) / dif->dz;

    aft->ta.Tzxx[idx] = (2.0 - Hzetadx * dif->dt) / (2.0 + Hzetadx * dif->dt) * bef->ta.Tzxx[idx]
        + 2.0 * (Hmu * dif->dt + Hgamma) / (2.0 + Hzetadx * dif->dt) * (aft->va.Vz[idx] - aft->va.Vz[idx_i1]) / dif->dx
        - 2.0 * Hgamma / (2.0 + Hzetadx * dif->dt) * (bef->va.Vz[idx] - bef->va.Vz[idx_i1]) / dif->dx;

  }
}

__global__ void DirectionalAddT(BefAft *aft, Range *ran, char check) {
  // スレッドインデックスの計算
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  int imax = ran->tr.Txy.x, jmax = ran->tr.Txy.y, kmax = ran->tr.Txy.z;

  if (i < imax && j < jmax && k < kmax) {
    int idx = k * imax * jmax + j * imax + i;

    if (check == 'X') {
        aft->ta.Tyz[idx] = aft->ta.Tyzy[idx] + aft->ta.Tyzz[idx];
    } else if (check == 'Y') {
        aft->ta.Tzx[idx] = aft->ta.Tzxx[idx] + aft->ta.Tzxz[idx];
    } else if (check == 'Z') {
        aft->ta.Txy[idx] = aft->ta.Txyx[idx] + aft->ta.Txyy[idx];
    } else {
        printf("error: DirectionalAddT");
    }

  }
}

// Txyクラス的な
void Txy(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Coord threads) {
  // hipError_t err;
  int Txyimax = ran_h->tr.Txy.x, Txyjmax = ran_h->tr.Txy.y, Txykmax = ran_h->tr.Txy.z;
  dim3 threadsPerBlock(threads.x, threads.y, threads.z); // 1ブロックあたりのスレッド数
  dim3 UpdateBlocks((Txyimax - 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (Txyjmax - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (Txykmax - 2 + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // dim3 ZeroXYBlocks((Txyimax - 1 + threadsPerBlock.x - 1) / threadsPerBlock.x, (Txyjmax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
  dim3 DirectionalAddBlocks((Txyimax + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (Txyjmax + threadsPerBlock.y - 1) / threadsPerBlock.y,
                            (Txykmax + threadsPerBlock.z - 1) / threadsPerBlock.z);                    
  TxyUpdate<<<UpdateBlocks, threadsPerBlock>>>(aft_d, bef_d, ma_d, dif_d, ran_d);
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Txy update: %s\n", hipGetErrorString(err));
  // ZeroTxy<<<ZeroXYBlocks, threadsPerBlock>>>(aft_d, ran_d);
  hipDeviceSynchronize();

  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Txy zero  : %s\n", hipGetErrorString(err));
  DirectionalAddT<<<DirectionalAddBlocks, threadsPerBlock>>>(aft_d, ran_d, 'Z');
  hipDeviceSynchronize();

  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Txy add   : %s\n", hipGetErrorString(err));
}
// Tyzクラス的な
void Tyz(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Coord threads) {
  // hipError_t err;
  int Tyzimax = ran_h->tr.Tyz.x, Tyzjmax = ran_h->tr.Tyz.y, Tyzkmax = ran_h->tr.Tyz.z;
  dim3 threadsPerBlock(threads.x, threads.y, threads.z); // 1ブロックあたりのスレッド数
  dim3 UpdateBlocks((Tyzimax - 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (Tyzjmax - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (Tyzkmax - 2 + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // dim3 ZeroYZBlocks((Tyzjmax - 1 + threadsPerBlock.x - 1) / threadsPerBlock.x, (Tyzkmax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
  dim3 DirectionalAddBlocks((Tyzimax + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (Tyzjmax + threadsPerBlock.y - 1) / threadsPerBlock.y,
                            (Tyzkmax + threadsPerBlock.z - 1) / threadsPerBlock.z);                    
  TyzUpdate<<<UpdateBlocks, threadsPerBlock>>>(aft_d, bef_d, ma_d, dif_d, ran_d);
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tyz update: %s\n", hipGetErrorString(err));
  // ZeroTyz<<<ZeroYZBlocks, threadsPerBlock>>>(aft_d, ran_d);
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tyz zero  : %s\n", hipGetErrorString(err));
  DirectionalAddT<<<DirectionalAddBlocks, threadsPerBlock>>>(aft_d, ran_d, 'X');
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tyz add   : %s\n", hipGetErrorString(err));
}
// Tzxクラス的な
void Tzx(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Coord threads) {
  // hipError_t err;
  int Tzximax = ran_h->tr.Tzx.x, Tzxjmax = ran_h->tr.Tzx.y, Tzxkmax = ran_h->tr.Tzx.z;
  dim3 threadsPerBlock(threads.x, threads.y, threads.z); // 1ブロックあたりのスレッド数
  dim3 UpdateBlocks((Tzximax - 2 + threadsPerBlock.x - 1)     / threadsPerBlock.x,
                    (Tzxjmax - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (Tzxkmax - 2 + threadsPerBlock.z - 1)     / threadsPerBlock.z);
  // dim3 ZeroZXBlocks((Tzxkmax - 1 + threadsPerBlock.x - 1) / threadsPerBlock.x, (Tzximax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);   
  dim3 DirectionalAddBlocks((Tzximax + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (Tzxjmax + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                            (Tzxkmax + threadsPerBlock.z - 1) / threadsPerBlock.z);                  
  TzxUpdate<<<UpdateBlocks, threadsPerBlock>>>(aft_d, bef_d, ma_d, dif_d, ran_d);
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tzx update: %s\n", hipGetErrorString(err));
  // ZeroTzx<<<ZeroZXBlocks, threadsPerBlock>>>(aft_d, ran_d);
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tzx zero  : %s\n", hipGetErrorString(err));
  DirectionalAddT<<<DirectionalAddBlocks, threadsPerBlock>>>(aft_d, ran_d, 'Y');
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Tzx add   : %s\n", hipGetErrorString(err));

}
// せん断応力計算(main呼び出し関数)
void Tau(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Coord threads) {
  Txy(aft_d, bef_d, ma_d, dif_d, ran_d, ran_h, threads);
  Tyz(aft_d, bef_d, ma_d, dif_d, ran_d, ran_h, threads);
  Tzx(aft_d, bef_d, ma_d, dif_d, ran_d, ran_h, threads);
}

// 粒子速度

// 粒子速度更新関数
__global__ void VxUpdate(BefAft *aft, BefAft *bef, MedArr *ma, Diff *dif, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
  int imax = ran->vr.Vx.x, jmax = ran->vr.Vx.y, kmax = ran->vr.Vx.z;
  double Azetaxx, Azetaxy, Azetaxz, Arho;
  // printf("%d,%d,%d\n", imax ,jmax ,kmax);


  if(i < imax - 1 && j < jmax - 1 && k < kmax - 1) {
    // 1D indexing for 3D arrays
    // printf("ok\n");
    int idx    = k * imax * jmax + j * imax + i;
    int idx_i1 = k * imax * jmax + j * imax + (i - 1);
    int idx_j1 = k * imax * jmax + (j + 1) * imax + i;
    int idx_k1 = (k + 1) * imax * jmax + j * imax + i;

    Azetaxx = (ma->zetaxx[idx_i1] + ma->zetaxx[idx]) / 2.;
    Azetaxy = (ma->zetaxy[idx_i1] + ma->zetaxy[idx]) / 2.;
    Azetaxz = (ma->zetaxz[idx_i1] + ma->zetaxz[idx]) / 2.;
    Arho    = (ma->rho[idx_i1] + ma->rho[idx]) / 2.;

    aft->va.Vxx[idx] = (2. * Arho - Azetaxx * dif->dt) / (2. * Arho + Azetaxx * dif->dt) * bef->va.Vxx[idx]
        + 2. * dif->dt / (2. * Arho + Azetaxx * dif->dt) * (bef->sa.Txx[idx] - bef->sa.Txx[idx_i1]) / dif->dx;

    aft->va.Vxy[idx] = (2. * Arho - Azetaxy * dif->dt) / (2. * Arho + Azetaxy * dif->dt) * bef->va.Vxy[idx]
        + 2. * dif->dt / (2. * Arho + Azetaxy * dif->dt) * (bef->ta.Txy[idx_j1] - bef->ta.Txy[idx]) / dif->dy;

    aft->va.Vxz[idx] = (2. * Arho - Azetaxz * dif->dt) / (2. * Arho + Azetaxz * dif->dt) * bef->va.Vxz[idx]
        + 2. * dif->dt / (2. * Arho + Azetaxz * dif->dt) * (bef->ta.Tzx[idx_k1] - bef->ta.Tzx[idx]) / dif->dz;

  }
}
// 粒子速度更新関数
__global__ void VyUpdate(BefAft *aft, BefAft *bef, MedArr *ma, Diff *dif, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

  int imax = ran->vr.Vy.x, jmax = ran->vr.Vy.y, kmax = ran->vr.Vy.z;
  double Azetayx, Azetayy, Azetayz, Arho;

  if (i < imax - 1 && j < jmax - 1 && k < kmax - 1) {
    // 各インデックスの計算
    int idx    = k * imax * jmax + j * imax + i;
    int idx_i1 = k * imax * jmax + j * imax + (i + 1);
    int idx_j1 = k * imax * jmax + (j - 1) * imax + i;
    int idx_k1 = (k + 1) * imax * jmax + j * imax + i;

    // 各種パラメータの計算
    Azetayx = (ma->zetayx[idx_j1] + ma->zetayx[idx]) / 2.0;
    Azetayy = (ma->zetayy[idx_j1] + ma->zetayy[idx]) / 2.0;
    Azetayz = (ma->zetayz[idx_j1] + ma->zetayz[idx]) / 2.0;
    Arho    = (ma->rho[idx_j1] + ma->rho[idx]) / 2.0;

    // Vyxの更新
    aft->va.Vyx[idx] = (2.0 * Arho - Azetayx * dif->dt) / (2.0 * Arho + Azetayx * dif->dt) * bef->va.Vyx[idx]
        + 2.0 * dif->dt / (2.0 * Arho + Azetayx * dif->dt) * (bef->ta.Txy[idx_i1] - bef->ta.Txy[idx]) / dif->dx;

    // Vyyの更新
    aft->va.Vyy[idx] = (2.0 * Arho - Azetayy * dif->dt) / (2.0 * Arho + Azetayy * dif->dt) * bef->va.Vyy[idx]
        + 2.0 * dif->dt / (2.0 * Arho + Azetayy * dif->dt) * (bef->sa.Tyy[idx] - bef->sa.Tyy[idx_j1]) / dif->dy;

    // Vyzの更新
    aft->va.Vyz[idx] = (2.0 * Arho - Azetayz * dif->dt) / (2.0 * Arho + Azetayz * dif->dt) * bef->va.Vyz[idx]
        + 2.0 * dif->dt / (2.0 * Arho + Azetayz * dif->dt) * (bef->ta.Tyz[idx_k1] - bef->ta.Tyz[idx]) / dif->dz;

  }
}
// 粒子速度更新関数
__global__ void VzUpdate(BefAft *aft, BefAft *bef, MedArr *ma, Diff *dif, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

  int imax = ran->vr.Vz.x, jmax = ran->vr.Vz.y, kmax = ran->vr.Vz.z;
  double Azetazx, Azetazy, Azetazz, Arho;

  if(i < imax - 1 && j < jmax - 1 && k < kmax - 1) {
    // 1D indexing for 3D arrays
    int idx    = k * imax * jmax + j * imax + i;
    int idx_i1 = k * imax * jmax + j * imax + (i + 1);
    int idx_j1 = k * imax * jmax + (j + 1) * imax + i;
    int idx_k1 = (k - 1) * imax * jmax + j * imax + i;

    Azetazx = (ma->zetazx[idx_k1] + ma->zetazx[idx]) / 2.;
    Azetazy = (ma->zetazy[idx_k1] + ma->zetazy[idx]) / 2.;
    Azetazz = (ma->zetazz[idx_k1] + ma->zetazz[idx]) / 2.;
    Arho    = (ma->rho[idx_k1] + ma->rho[idx]) / 2.;
    aft->va.Vzx[idx] = (2. * Arho - Azetazx * dif->dt) / (2. * Arho + Azetazx * dif->dt) * bef->va.Vzx[idx]
        + 2. * dif->dt / (2. * Arho + Azetazx * dif->dt) * (bef->ta.Tzx[idx_i1] - bef->ta.Tzx[idx]) / dif->dx;

    aft->va.Vzy[idx] = (2. * Arho - Azetazy * dif->dt) / (2. * Arho + Azetazy * dif->dt) * bef->va.Vzy[idx]
        + 2. * dif->dt / (2. * Arho + Azetazy * dif->dt) * (bef->ta.Tyz[idx_j1] - bef->ta.Tyz[idx]) / dif->dy;

    aft->va.Vzz[idx] = (2. * Arho - Azetazz * dif->dt) / (2. * Arho + Azetazz * dif->dt) * bef->va.Vzz[idx]
        + 2. * dif->dt / (2. * Arho + Azetazz * dif->dt) * (bef->sa.Tzz[idx] - bef->sa.Tzz[idx_k1]) / dif->dz;
  }
}

__global__ void DirectionalAddV(BefAft *aft, Range *ran, char check) {
  // スレッドインデックスの計算
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  int imax = ran->vr.Vx.x, jmax = ran->vr.Vx.y, kmax = ran->vr.Vx.z;


  if (i < imax && j < jmax && k < kmax) {
    int idx = k * imax * jmax + j * imax + i;

    if (check == 'X') {
        aft->va.Vx[idx] = aft->va.Vxx[idx] + aft->va.Vxy[idx] + aft->va.Vxz[idx];
    } else if (check == 'Y') {
        aft->va.Vy[idx] = aft->va.Vyx[idx] + aft->va.Vyy[idx] + aft->va.Vyz[idx];
    } else if (check == 'Z') {
        aft->va.Vz[idx] = aft->va.Vzx[idx] + aft->va.Vzy[idx] + aft->va.Vzz[idx];
    } else {
      printf("error: DirectionalAddV");
    }
  }
}

// Vxクラス的な
void Vx(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Coord threads) {
  int Vximax = ran_h->vr.Vx.x, Vxjmax = ran_h->vr.Vx.y, Vxkmax = ran_h->vr.Vx.z;
  dim3 threadsPerBlock(threads.x, threads.y, threads.z); // 1ブロックあたりのスレッド数
  dim3 UpdateBlocks((Vximax - 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (Vxjmax - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (Vxkmax - 2 + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // dim3 ZeroXYBlocks((Vximax + threadsPerBlock.x - 1) / threadsPerBlock.x, 
  //                   (Vxjmax - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // dim3 ZeroXZBlocks((Vximax + threadsPerBlock.x - 1) / threadsPerBlock.x, 
  //                   (Vxkmax + threadsPerBlock.y - 1) / threadsPerBlock.y);  
  dim3 DirectionalAddBlocks((Vximax + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (Vxjmax + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                            (Vxkmax + threadsPerBlock.z - 1) / threadsPerBlock.z);
  VxUpdate<<<UpdateBlocks, threadsPerBlock>>>(aft_d, bef_d, ma_d, dif_d, ran_d);
  // hipError_t err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Vx  update: %s\n", hipGetErrorString(err));
  // ZeroVx_XY<<<ZeroXYBlocks, threadsPerBlock>>>(aft_d, ran_d);
  // ZeroVx_XZ<<<ZeroXZBlocks, threadsPerBlock>>>(aft_d, ran_d);
  
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Vx  zero  : %s\n", hipGetErrorString(err));
  DirectionalAddV<<<DirectionalAddBlocks, threadsPerBlock>>>(aft_d, ran_d, 'X');
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Vx  add   : %s\n", hipGetErrorString(err));

}
// Vyクラス的な
void Vy(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Coord threads) {

  int Vyimax = ran_h->vr.Vy.x, Vyjmax = ran_h->vr.Vy.y, Vykmax = ran_h->vr.Vy.z;

  dim3 threadsPerBlock(threads.x, threads.y, threads.z); // 1ブロックあたりのスレッド数
  dim3 UpdateBlocks((Vyimax - 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (Vyjmax - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (Vykmax - 2 + threadsPerBlock.z - 1) / threadsPerBlock.z);
  // dim3 ZeroYXBlocks((Vyimax - 1 + threadsPerBlock.x - 1) / threadsPerBlock.x, 
  //                   (Vyjmax + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // dim3 ZeroYZBlocks((Vyjmax + threadsPerBlock.x - 1) / threadsPerBlock.x, 
  //                   (Vykmax + threadsPerBlock.y - 1) / threadsPerBlock.y);  
  dim3 DirectionalAddBlocks((Vyimax + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (Vyjmax + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                            (Vykmax + threadsPerBlock.z - 1) / threadsPerBlock.z);
  VyUpdate<<<UpdateBlocks, threadsPerBlock>>>(aft_d, bef_d, ma_d, dif_d, ran_d);
  // hipError_t err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Vy  update: %s\n", hipGetErrorString(err));
  // ZeroVy_YX<<<ZeroYXBlocks, threadsPerBlock>>>(aft_d, ran_d);
  // ZeroVy_YZ<<<ZeroYZBlocks, threadsPerBlock>>>(aft_d, ran_d);
 
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Vy  zero  : %s\n", hipGetErrorString(err));

  //全方向加算
  DirectionalAddV<<<DirectionalAddBlocks, threadsPerBlock>>>(aft_d, ran_d, 'Y');
  hipDeviceSynchronize();
  hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Vy  add   : %s\n", hipGetErrorString(err));

}
// Vzクラス的な
void Vz(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Coord threads) {

  int Vzimax = ran_h->vr.Vz.x, Vzjmax = ran_h->vr.Vz.y, Vzkmax = ran_h->vr.Vz.z;

  dim3 threadsPerBlock(threads.x, threads.y, threads.z); // 1ブロックあたりのスレッド数
  dim3 UpdateBlocks((Vzimax - 2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (Vzjmax - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (Vzkmax - 2 + threadsPerBlock.z - 1)     / threadsPerBlock.z);
  // dim3 ZeroZXBlocks((Vzimax - 1 + threadsPerBlock.x - 1) / threadsPerBlock.x, 
  //                   (Vzkmax + threadsPerBlock.y - 1) / threadsPerBlock.y);
  // dim3 ZeroZYBlocks((Vzjmax + threadsPerBlock.x - 1) / threadsPerBlock.x, 
  //                   (Vzkmax + threadsPerBlock.y - 1) / threadsPerBlock.y);  
  dim3 DirectionalAddBlocks((Vzimax + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (Vzjmax + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                            (Vzkmax + threadsPerBlock.z - 1) / threadsPerBlock.z);                    
  VzUpdate<<<UpdateBlocks, threadsPerBlock>>>(aft_d, bef_d, ma_d, dif_d, ran_d);

  // hipError_t err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Vz  update: %s\n", hipGetErrorString(err));
  // ZeroVz_ZX<<<ZeroZXBlocks, threadsPerBlock>>>(aft_d, ran_d);
  // ZeroVz_ZY<<<ZeroZYBlocks, threadsPerBlock>>>(aft_d, ran_d);
 
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Vz  zero  : %s\n", hipGetErrorString(err));
  //全方向加算
  DirectionalAddV<<<DirectionalAddBlocks, threadsPerBlock>>>(aft_d, ran_d, 'Z');
  hipDeviceSynchronize();
  // err = hipGetLastError(); // カーネル呼び出し後にエラーチェック
  // printf("CUDA kernel error Vz  add   : %s\n", hipGetErrorString(err));

}
//粒子速度計算
void Vel(BefAft *aft_d, BefAft *bef_d, MedArr *ma_d, Diff *dif_d, Range *ran_d, Range *ran_h, Coord threads) {
  Vx(aft_d, bef_d, ma_d, dif_d, ran_d, ran_h, threads);
  Vy(aft_d, bef_d, ma_d, dif_d, ran_d, ran_h, threads);
  Vz(aft_d, bef_d, ma_d, dif_d, ran_d, ran_h, threads);
}

//更新
__global__ void swapTxx(BefAft *aft, BefAft *bef, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int Txximax = ran->sr.Txx.x, Txxjmax = ran->sr.Txx.y, Txxkmax = ran->sr.Txx.z;
  if (i < Txximax && j < Txxjmax && k < Txxkmax) {
    int idx_Txx = k * Txximax * Txxjmax + j * Txximax + i;
    *(bef->sa.Txx  + idx_Txx) = *(aft->sa.Txx  + idx_Txx);
    *(bef->sa.Txxx + idx_Txx) = *(aft->sa.Txxx + idx_Txx);
    *(bef->sa.Txxy + idx_Txx) = *(aft->sa.Txxy + idx_Txx);
    *(bef->sa.Txxz + idx_Txx) = *(aft->sa.Txxz + idx_Txx);
  }
}

__global__ void swapTyy(BefAft *aft, BefAft *bef, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int Tyyimax = ran->sr.Tyy.x, Tyyjmax = ran->sr.Tyy.y, Tyykmax = ran->sr.Tyy.z;
  
  if (i < Tyyimax && j < Tyyjmax && k < Tyykmax) {
    int idx_Tyy = k * Tyyimax * Tyyjmax + j * Tyyimax + i;
    *(bef->sa.Tyy  + idx_Tyy) = *(aft->sa.Tyy  + idx_Tyy);
    *(bef->sa.Tyyx + idx_Tyy) = *(aft->sa.Tyyx + idx_Tyy);
    *(bef->sa.Tyyy + idx_Tyy) = *(aft->sa.Tyyy + idx_Tyy);
    *(bef->sa.Tyyz + idx_Tyy) = *(aft->sa.Tyyz + idx_Tyy);
  }
}

__global__ void swapTzz(BefAft *aft, BefAft *bef, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int Tzzimax = ran->sr.Tzz.x, Tzzjmax = ran->sr.Tzz.y, Tzzkmax = ran->sr.Tzz.z;

  if (i < Tzzimax && j < Tzzjmax && k < Tzzkmax) {
    int idx_Tzz = k * Tzzimax * Tzzjmax + j * Tzzimax + i;
    *(bef->sa.Tzz  + idx_Tzz) = *(aft->sa.Tzz  + idx_Tzz);
    *(bef->sa.Tzzx + idx_Tzz) = *(aft->sa.Tzzx + idx_Tzz);
    *(bef->sa.Tzzy + idx_Tzz) = *(aft->sa.Tzzy + idx_Tzz);
    *(bef->sa.Tzzz + idx_Tzz) = *(aft->sa.Tzzz + idx_Tzz);
  }
}

__global__ void swapTxy(BefAft *aft, BefAft *bef, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int Txyimax = ran->tr.Txy.x, Txyjmax = ran->tr.Txy.y, Txykmax = ran->tr.Txy.z;

  if (i < Txyimax && j < Txyjmax && k < Txykmax) {
    int idx_Txy = k * Txyimax * Txyjmax + j * Txyimax + i;
    *(bef->ta.Txy  + idx_Txy) = *(aft->ta.Txy  + idx_Txy);
    *(bef->ta.Txyx + idx_Txy) = *(aft->ta.Txyx + idx_Txy);
    *(bef->ta.Txyy + idx_Txy) = *(aft->ta.Txyy + idx_Txy);
  }
}

__global__ void swapTyz(BefAft *aft, BefAft *bef, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int Tyzimax = ran->tr.Tyz.x, Tyzjmax = ran->tr.Tyz.y, Tyzkmax = ran->tr.Tyz.z;

  if (i < Tyzimax && j < Tyzjmax && k < Tyzkmax) {
    int idx_Tyz = k * Tyzimax * Tyzjmax + j * Tyzimax + i;
    *(bef->ta.Tyz  + idx_Tyz) = *(aft->ta.Tyz  + idx_Tyz);
    *(bef->ta.Tyzy + idx_Tyz) = *(aft->ta.Tyzy + idx_Tyz);
    *(bef->ta.Tyzz + idx_Tyz) = *(aft->ta.Tyzz + idx_Tyz);
  }
}

__global__ void swapTzx(BefAft *aft, BefAft *bef, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int Tzximax = ran->tr.Tzx.x, Tzxjmax = ran->tr.Tzx.y, Tzxkmax = ran->tr.Tzx.z;
  

  if (i < Tzximax && j < Tzxjmax && k < Tzxkmax) {
    int idx_Tzx = k * Tzximax * Tzxjmax + j * Tzximax + i;
    *(bef->ta.Tzx  + idx_Tzx) = *(aft->ta.Tzx  + idx_Tzx);
    *(bef->ta.Tzxz + idx_Tzx) = *(aft->ta.Tzxz + idx_Tzx);
    *(bef->ta.Tzxx + idx_Tzx) = *(aft->ta.Tzxx + idx_Tzx);
  }
}

__global__ void swapVx(BefAft *aft, BefAft *bef, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int Vximax = ran->vr.Vx.x, Vxjmax = ran->vr.Vx.y, Vxkmax = ran->vr.Vx.z;
  
  if (i < Vximax && j < Vxjmax && k < Vxkmax) {
    int idx_Vx = k * Vximax * Vxjmax + j * Vximax + i;
    *(bef->va.Vx  + idx_Vx) = *(aft->va.Vx  + idx_Vx);
    *(bef->va.Vxx + idx_Vx) = *(aft->va.Vxx + idx_Vx);
    *(bef->va.Vxy + idx_Vx) = *(aft->va.Vxy + idx_Vx);
    *(bef->va.Vxz + idx_Vx) = *(aft->va.Vxz + idx_Vx);
  }
}

__global__ void swapVy(BefAft *aft, BefAft *bef, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int Vyimax = ran->vr.Vy.x, Vyjmax = ran->vr.Vy.y, Vykmax = ran->vr.Vy.z;
  

  if (i < Vyimax && j < Vyjmax && k < Vykmax) {
    int idx_Vy = k * Vyimax * Vyjmax + j * Vyimax + i;
    *(bef->va.Vy  + idx_Vy) = *(aft->va.Vy  + idx_Vy);
    *(bef->va.Vyx + idx_Vy) = *(aft->va.Vyx + idx_Vy);
    *(bef->va.Vyy + idx_Vy) = *(aft->va.Vyy + idx_Vy);
    *(bef->va.Vyz + idx_Vy) = *(aft->va.Vyz + idx_Vy);
  }
}

__global__ void swapVz(BefAft *aft, BefAft *bef, Range *ran) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int Vzimax = ran->vr.Vz.x, Vzjmax = ran->vr.Vz.y, Vzkmax = ran->vr.Vz.z;
  

  if (i < Vzimax && j < Vzjmax && k < Vzkmax) {
    int idx_Vz = k * Vzimax * Vzjmax + j * Vzimax + i;
    *(bef->va.Vz  + idx_Vz) = *(aft->va.Vz  + idx_Vz);
    *(bef->va.Vzx + idx_Vz) = *(aft->va.Vzx + idx_Vz);
    *(bef->va.Vzy + idx_Vz) = *(aft->va.Vzy + idx_Vz);
    *(bef->va.Vzz + idx_Vz) = *(aft->va.Vzz + idx_Vz);
  }
}

void swapBefAft(BefAft *aft, BefAft *bef, Range *ran_h, Range *ran_d, Coord threads) {
  dim3 threadsPerBlock(threads.x, threads.y, threads.z); // 1ブロックあたりのスレッド数
  dim3 SwapTxxBlocks((ran_h->sr.Txx.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h->sr.Txx.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h->sr.Txx.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 SwapTyyBlocks((ran_h->sr.Tyy.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h->sr.Tyy.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h->sr.Tyy.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 SwapTzzBlocks((ran_h->sr.Tzz.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h->sr.Tzz.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h->sr.Tzz.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 SwapTxyBlocks((ran_h->tr.Txy.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h->tr.Txy.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h->tr.Txy.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 SwapTyzBlocks((ran_h->tr.Tyz.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h->tr.Tyz.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h->tr.Tyz.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 SwapTzxBlocks((ran_h->tr.Tzx.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h->tr.Tzx.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h->tr.Tzx.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3  SwapVxBlocks((ran_h->vr.Vx.x  + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h->vr.Vx.y  + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h->vr.Vx.z  + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3  SwapVyBlocks((ran_h->vr.Vy.x  + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h->vr.Vy.y  + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h->vr.Vy.z  + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3  SwapVzBlocks((ran_h->vr.Vz.x  + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h->vr.Vz.y  + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h->vr.Vz.z  + threadsPerBlock.z - 1) / threadsPerBlock.z);
  swapTxx<<<SwapTxxBlocks, threadsPerBlock>>>(aft, bef, ran_d);
  swapTyy<<<SwapTyyBlocks, threadsPerBlock>>>(aft, bef, ran_d);
  swapTzz<<<SwapTzzBlocks, threadsPerBlock>>>(aft, bef, ran_d);
  swapTxy<<<SwapTxyBlocks, threadsPerBlock>>>(aft, bef, ran_d);
  swapTyz<<<SwapTyzBlocks, threadsPerBlock>>>(aft, bef, ran_d);
  swapTzx<<<SwapTzxBlocks, threadsPerBlock>>>(aft, bef, ran_d);
  swapVx<<<SwapVxBlocks, threadsPerBlock>>>(aft, bef, ran_d);
  swapVy<<<SwapVyBlocks, threadsPerBlock>>>(aft, bef, ran_d);
  swapVz<<<SwapVzBlocks, threadsPerBlock>>>(aft, bef, ran_d);
  hipDeviceSynchronize();
}
