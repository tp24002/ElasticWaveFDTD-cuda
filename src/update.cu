#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include "../header/update.h"

#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

#include "../header/struct.h"

__device__ void Txx(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, SigRan sr, Inpaluse ip, int t) {
  int i, j, k;
  int Txximax = sr.Txx.x, Txxjmax = sr.Txx.y, Txxkmax = sr.Txx.z;
  if (ip.mode == E_SINE) {
    ip.Txx[ip.in.x][ip.in.y][ip.in.z] = 0;
  } else if (ip.mode == E_RCOS) {
    if (t < 1. / ip.freq / dif.dt) {
      ip.Txx[ip.in.x][ip.in.y][ip.in.z] = 0;///* 8.e3 * 0.5 * */(1. - cos(2. * M_PI * ip.freq * (double)t * dif.dt)) / 2.;
    } else {
      ip.Txx[ip.in.x][ip.in.y][ip.in.z] = 0.;
    }
  } else {
    ip.Txx[ip.in.x][ip.in.y][ip.in.z] = 0.;
  }
#pragma omp parallel for private(i, j, k)
  for (k = 1; k <= Txxkmax - 1; k++) {
    for (j = 1; j <= Txxjmax - 1; j++) {
      for (i = 1; i <= Txximax - 1; i++) {
        aft->sa.Txxx[i][j][k] = (2. - ma.zetadx[i][j][k] * dif.dt) / (2. + ma.zetadx[i][j][k] * dif.dt) * bef->sa.Txxx[i][j][k]
         + 2. * (ma.c11[i][j][k] * dif.dt + ma.xi11[i][j][k]) / (2. + ma.zetadx[i][j][k] * dif.dt) * (aft->va.Vx[i][j][k] - aft->va.Vx[i - 1][j][k]) / dif.dx 
          - 2. * ma.xi11[i][j][k] / (2. + ma.zetadx[i][j][k] * dif.dt) * (bef->va.Vx[i][j][k] - bef->va.Vx[i - 1][j][k]) / dif.dx;

        aft->sa.Txxy[i][j][k] = (2. - ma.zetady[i][j][k] * dif.dt) / (2. + ma.zetady[i][j][k] * dif.dt) * bef->sa.Txxy[i][j][k]
         + 2. * (ma.ramda[i][j][k] * dif.dt + ma.khi[i][j][k]) / (2. + ma.zetady[i][j][k] * dif.dt) * (aft->va.Vy[i][j][k] - aft->va.Vy[i][j - 1][k]) / dif.dy
          - 2. * ma.khi[i][j][k] / (2. + ma.zetady[i][j][k] * dif.dt) * (bef->va.Vy[i][j][k] - bef->va.Vy[i][j - 1][k]) / dif.dy;

        aft->sa.Txxz[i][j][k] = (2. - ma.zetadz[i][j][k] * dif.dt) / (2. + ma.zetadz[i][j][k] * dif.dt) * bef->sa.Txxz[i][j][k]
         + 2. * (ma.ramda[i][j][k] * dif.dt + ma.khi[i][j][k]) / (2. + ma.zetadz[i][j][k] * dif.dt) * (aft->va.Vz[i][j][k] - aft->va.Vz[i][j][k - 1]) / dif.dz
          - 2. * ma.khi[i][j][k] / (2. + ma.zetadz[i][j][k] * dif.dt) * (bef->va.Vz[i][j][k] - bef->va.Vz[i][j][k - 1]) / dif.dz;
      }
    }
  }
  // // ここから
#pragma omp parallel for private(i, j)
  for (j = 0; j <= Txxjmax; j++) {
    for (i = 0; i <= Txximax; i++) {
      aft->sa.Txxx[i][j][0] = 0.;
      aft->sa.Txxx[i][j][Txxkmax] = 0.;
      aft->sa.Txxy[i][j][0] = 0.;
      aft->sa.Txxy[i][j][Txxkmax] = 0.;
      aft->sa.Txxz[i][j][0] = 0.;
      aft->sa.Txxz[i][j][Txxkmax] = 0.;
    }
  }
#pragma omp parallel for private(j, k)
  for (k = 1; k <= Txxkmax - 1; k++) {
    for (j = 0; j <= Txxjmax; j++) {
      aft->sa.Txxx[0][j][k] = 0.;
      aft->sa.Txxx[Txximax][j][k] = 0.;
      aft->sa.Txxy[0][j][k] = 0.;
      aft->sa.Txxy[Txximax][j][k] = 0.;
      aft->sa.Txxz[0][j][k] = 0.;
      aft->sa.Txxz[Txximax][j][k] = 0.;
    }
  }
#pragma omp parallel for private(i, k)
  for (k = 1; k <= Txxkmax - 1; k++) {
    for (i = 1; i <= Txximax - 1; i++) {
      aft->sa.Txxx[i][0][k] = 0.;
      aft->sa.Txxx[i][Txxjmax][k] = 0.;
      aft->sa.Txxy[i][0][k] = 0.;
      aft->sa.Txxy[i][Txxjmax][k] = 0.;
      aft->sa.Txxz[i][0][k] = 0.;
      aft->sa.Txxz[i][Txxjmax][k] = 0.;
    }
  }
//全方向加算
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Txxkmax; k++) {
    for (j = 0; j <= Txxjmax; j++) {
      for (i = 0; i <= Txximax; i++) {
        aft->sa.Txx[i][j][k] = aft->sa.Txxx[i][j][k] + aft->sa.Txxy[i][j][k] + aft->sa.Txxz[i][j][k] + ip.Txx[i][j][k];
      }
    }
  }
}

__device__ void Tyy(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, SigRan sr, Inpaluse ip, int t) {
  int i, j, k;
  int Tyyimax = sr.Tyy.x, Tyyjmax = sr.Tyy.y, Tyykmax = sr.Tyy.z;
  if (ip.mode == E_SINE) {
    ip.Tyy[ip.in.x][ip.in.y][ip.in.z] = 0;
  } else if (ip.mode == E_RCOS) {
    if (t < 1. / ip.freq / dif.dt) {
      ip.Tyy[ip.in.x][ip.in.y][ip.in.z] = 0;///* 8.e3 * 0.5 * */(1. - cos(2. * M_PI * ip.freq * (double)t * dif.dt)) / 2.;
    } else {
      ip.Tyy[ip.in.x][ip.in.y][ip.in.z] = 0.;
    }
  } else {
    ip.Tyy[ip.in.x][ip.in.y][ip.in.z] = 0.;
  }
#pragma omp parallel for private(i, j, k)
  for (k = 1; k <= Tyykmax - 1; k++) {
    for (j = 1; j <= Tyyjmax - 1; j++) {
      for (i = 1; i <= Tyyimax - 1; i++) {
        aft->sa.Tyyx[i][j][k] = (2. - ma.zetadx[i][j][k] * dif.dt) / (2. + ma.zetadx[i][j][k] * dif.dt) * bef->sa.Tyyx[i][j][k]
         + 2. * (ma.ramda[i][j][k] * dif.dt + ma.khi[i][j][k]) / (2. + ma.zetadx[i][j][k] * dif.dt) * (aft->va.Vx[i][j][k] - aft->va.Vx[i - 1][j][k]) / dif.dx
          - 2. * ma.khi[i][j][k] / (2. + ma.zetadx[i][j][k] * dif.dt) * (bef->va.Vx[i][j][k] - bef->va.Vx[i - 1][j][k]) / dif.dx;

        aft->sa.Tyyy[i][j][k] = (2. - ma.zetady[i][j][k] * dif.dt) / (2. + ma.zetady[i][j][k] * dif.dt) * bef->sa.Tyyy[i][j][k]
         + 2. * (ma.c11[i][j][k] * dif.dt + ma.xi11[i][j][k]) / (2. + ma.zetady[i][j][k] * dif.dt) * (aft->va.Vy[i][j][k] - aft->va.Vy[i][j - 1][k]) / dif.dy
          - 2. * ma.xi11[i][j][k] / (2. + ma.zetady[i][j][k] * dif.dt) * (bef->va.Vy[i][j][k] - bef->va.Vy[i][j - 1][k]) / dif.dy;

        aft->sa.Tyyz[i][j][k] = (2. - ma.zetadz[i][j][k] * dif.dt) / (2. + ma.zetadz[i][j][k] * dif.dt) * bef->sa.Tyyz[i][j][k]
         + 2. * (ma.ramda[i][j][k] * dif.dt + ma.khi[i][j][k]) / (2. + ma.zetadz[i][j][k] * dif.dt) * (aft->va.Vz[i][j][k] - aft->va.Vz[i][j][k - 1]) / dif.dz
          - 2. * ma.khi[i][j][k] / (2. + ma.zetadz[i][j][k] * dif.dt)  * (bef->va.Vz[i][j][k] - bef->va.Vz[i][j][k - 1]) / dif.dz;
      }
    }
  }
#pragma omp parallel for private(i, j)
  for (j = 0; j <= Tyyjmax; j++) {
    for (i = 0; i <= Tyyimax; i++) {
      aft->sa.Tyyx[i][j][0] = 0.;
      aft->sa.Tyyx[i][j][Tyykmax] = 0.;
      aft->sa.Tyyy[i][j][0] = 0.;
      aft->sa.Tyyy[i][j][Tyykmax] = 0.;
      aft->sa.Tyyz[i][j][0] = 0.;
      aft->sa.Tyyz[i][j][Tyykmax] = 0.;
    }
  }
#pragma omp parallel for private(j, k)
  for (k = 1; k <= Tyykmax - 1; k++) {
    for (j = 0; j <= Tyyjmax; j++) {
      aft->sa.Tyyx[0][j][k] = 0.;
      aft->sa.Tyyx[Tyyimax][j][k] = 0.;
      aft->sa.Tyyy[0][j][k] = 0.;
      aft->sa.Tyyy[Tyyimax][j][k] = 0.;
      aft->sa.Tyyz[0][j][k] = 0.;
      aft->sa.Tyyz[Tyyimax][j][k] = 0.;
    }
  }
#pragma omp parallel for private(i, k)
  for (k = 1; k <= Tyykmax - 1; k++) {
    for (i = 1; i <= Tyyimax - 1; i++) {
      aft->sa.Tyyx[i][0][k] = 0.;
      aft->sa.Tyyx[i][Tyyjmax][k] = 0.;
      aft->sa.Tyyy[i][0][k] = 0.;
      aft->sa.Tyyy[i][Tyyjmax][k] = 0.;
      aft->sa.Tyyz[i][0][k] = 0.;
      aft->sa.Tyyz[i][Tyyjmax][k] = 0.;
    }
  }
//全方向加算
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Tyykmax; k++) {
    for (j = 0; j <= Tyyjmax; j++) {
      for (i = 0; i <= Tyyimax; i++) {
        aft->sa.Tyy[i][j][k] = aft->sa.Tyyx[i][j][k] + aft->sa.Tyyy[i][j][k] + aft->sa.Tyyz[i][j][k] + ip.Tyy[i][j][k];
      }
    }
  }
}

__device__ void Tzz(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, SigRan sr, Inpaluse ip, int t) {
  int i, j, k;
  int Tzzimax = sr.Tzz.x, Tzzjmax = sr.Tzz.y, Tzzkmax = sr.Tzz.z;
  if (ip.mode == E_SINE) {
    ip.Tzz[ip.in.x][ip.in.y][ip.in.z] = (-1) * sin(2. * M_PI * ip.freq * (double)t * dif.dt) / 2.;
  } else if (ip.mode == E_RCOS) {
    if (t < 1. / ip.freq / dif.dt) {
      ip.Tzz[ip.in.x][ip.in.y][ip.in.z] = 8.e3 * 0.5 * (-1) * (1. - cos(2. * M_PI * ip.freq * (double)t * dif.dt)) / 2.;
    } else {
      ip.Tzz[ip.in.x][ip.in.y][ip.in.z] = 0.;
    }
  } else {
    ip.Tzz[ip.in.x][ip.in.y][ip.in.z] = 0.;
  }

  // Tzzの更新式
#pragma omp parallel for private(i, j, k)
  for (k = 1; k <= Tzzkmax - 1; k++) {
    for (j = 1; j <= Tzzjmax - 1; j++) {
      for (i = 1; i <= Tzzimax - 1; i++) {
        aft->sa.Tzzx[i][j][k] = (2. - ma.zetadx[i][j][k] * dif.dt) / (2. + ma.zetadx[i][j][k] * dif.dt) * bef->sa.Tzzx[i][j][k]
         + 2. * (ma.ramda[i][j][k] * dif.dt + ma.khi[i][j][k]) / (2. + ma.zetadx[i][j][k] * dif.dt) * (aft->va.Vx[i][j][k] - aft->va.Vx[i - 1][j][k]) / dif.dx
          - 2. * ma.khi[i][j][k] / (2. + ma.zetadx[i][j][k] * dif.dt) * (bef->va.Vx[i][j][k] - bef->va.Vx[i - 1][j][k]) / dif.dx;

        aft->sa.Tzzy[i][j][k] = (2. - ma.zetady[i][j][k] * dif.dt) / (2. + ma.zetady[i][j][k] * dif.dt) * bef->sa.Tzzy[i][j][k]
         + 2. * (ma.ramda[i][j][k] * dif.dt + ma.khi[i][j][k]) / (2. + ma.zetady[i][j][k] * dif.dt) * (aft->va.Vy[i][j][k] - aft->va.Vy[i][j - 1][k]) / dif.dy
          - 2. * ma.khi[i][j][k] / (2. + ma.zetady[i][j][k] * dif.dt)  * (bef->va.Vy[i][j][k] - bef->va.Vy[i][j - 1][k]) / dif.dy;

        aft->sa.Tzzz[i][j][k] = (2. - ma.zetadz[i][j][k] * dif.dt) / (2. + ma.zetadz[i][j][k] * dif.dt) * bef->sa.Tzzz[i][j][k]
         + 2. * (ma.c11[i][j][k] * dif.dt + ma.xi11[i][j][k]) / (2. + ma.zetadz[i][j][k] * dif.dt) * (aft->va.Vz[i][j][k] - aft->va.Vz[i][j][k - 1]) / dif.dz
          - 2. * ma.xi11[i][j][k] / (2. + ma.zetadz[i][j][k] * dif.dt) * (bef->va.Vz[i][j][k] - bef->va.Vz[i][j][k - 1]) / dif.dz;
      }
    }
  }
#pragma omp parallel for private(i, j)
  for (j = 0; j <= Tzzjmax; j++) {
    for (i = 0; i <= Tzzimax; i++) {
      aft->sa.Tzzx[i][j][0] = 0.;
      aft->sa.Tzzx[i][j][Tzzkmax] = 0.;
      aft->sa.Tzzy[i][j][0] = 0.;
      aft->sa.Tzzy[i][j][Tzzkmax] = 0.;
      aft->sa.Tzzz[i][j][0] = 0.;
      aft->sa.Tzzz[i][j][Tzzkmax] = 0.;
    }
  }
#pragma omp parallel for private(j, k)
  for (k = 1; k <= Tzzkmax - 1; k++) {
    for (j = 0; j <= Tzzjmax; j++) {
      aft->sa.Tzzx[0][j][k] = 0.;
      aft->sa.Tzzx[Tzzimax][j][k] = 0.;
      aft->sa.Tzzy[0][j][k] = 0.;
      aft->sa.Tzzy[Tzzimax][j][k] = 0.;
      aft->sa.Tzzz[0][j][k] = 0.;
      aft->sa.Tzzz[Tzzimax][j][k] = 0.;
    }
  }
#pragma omp parallel for private(i, k)
  for (k = 1; k <= Tzzkmax - 1; k++) {
    for (i = 1; i <= Tzzimax - 1; i++) {
      aft->sa.Tzzx[i][0][k] = 0.;
      aft->sa.Tzzx[i][Tzzjmax][k] = 0.;
      aft->sa.Tzzy[i][0][k] = 0.;
      aft->sa.Tzzy[i][Tzzjmax][k] = 0.;
      aft->sa.Tzzz[i][0][k] = 0.;
      aft->sa.Tzzz[i][Tzzjmax][k] = 0.;
    }
  }
//全方向加算
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Tzzkmax; k++) {
    for (j = 0; j <= Tzzjmax; j++) {
      for (i = 0; i <= Tzzimax; i++) {
        aft->sa.Tzz[i][j][k] = aft->sa.Tzzx[i][j][k] + aft->sa.Tzzy[i][j][k] + aft->sa.Tzzz[i][j][k] + ip.Tzz[i][j][k];
      }
    }
  }
}
//垂直応力計算
__global__ void Sig(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, SigRan sr, Inpaluse ip, int t) {
  Txx(aft, bef, ma, dif, sr, ip, t);
  Tyy(aft, bef, ma, dif, sr, ip, t);
  Tzz(aft, bef, ma, dif, sr, ip, t);
}

void Txy(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, TauRan tr) {
  int i, j, k;
  int Txyimax = tr.Txy.x, Txyjmax = tr.Txy.y, Txykmax = tr.Txy.z;
  double Hzetadx, Hzetady, Hzetadz, Hmu, Hgamma;
#pragma omp parallel for private(i, j, k)
  for (k = 1; k <= Txykmax - 1; k++) {
    for (j = 0; j <= Txyjmax; j++) {
      for (i = 0; i <= Txyimax; i++) {
        //PML:減衰係数,計算領域:摩擦定数
        Hzetadx = 4. * pow((1. / ma.zetadx[i + 1][j + 1][k]) + (1. / ma.zetadx[i][j + 1][k]) + (1. / ma.zetadx[i + 1][j][k]) + (1. / ma.zetadx[i][j][k]), -1.);
        //PML:減衰係数,計算領域:摩擦定数
        Hzetady = 4. * pow((1. / ma.zetady[i + 1][j + 1][k]) + (1. / ma.zetady[i][j + 1][k]) + (1. / ma.zetady[i + 1][j][k]) + (1. / ma.zetady[i][j][k]), -1.);
        //第2ラメ，横弾性係数(剛性率)
        Hmu     = 4. * pow((1. /     ma.mu[i + 1][j + 1][k]) + (1. /     ma.mu[i][j + 1][k]) + (1. /     ma.mu[i + 1][j][k]) + (1. /     ma.mu[i][j][k]), -1.);
        //第１粘性定数
        Hgamma  = 4. * pow((1. /  ma.gamma[i + 1][j + 1][k]) + (1. /  ma.gamma[i][j + 1][k]) + (1. /  ma.gamma[i + 1][j][k]) + (1. /  ma.gamma[i][j][k]), -1.);
        aft->ta.Txyx[i][j][k] = (2. - Hzetadx * dif.dt) / (2. + Hzetadx * dif.dt) * bef->ta.Txyx[i][j][k]
         + 2. * (Hmu * dif.dt + Hgamma) / (2. + Hzetadx * dif.dt) * (aft->va.Vy[i + 1][j][k] - aft->va.Vy[i][j][k]) / dif.dx
          - 2. * Hgamma / (2. + Hzetadx * dif.dt) * (bef->va.Vy[i + 1][j][k] - bef->va.Vy[i][j][k]) / dif.dx;

        aft->ta.Txyy[i][j][k] = (2. - Hzetady * dif.dt) / (2. + Hzetady * dif.dt) * bef->ta.Txyy[i][j][k]
         + 2. * (Hmu * dif.dt + Hgamma) / (2. + Hzetady * dif.dt) * (aft->va.Vx[i][j + 1][k] - aft->va.Vx[i][j][k]) / dif.dy
          - 2. * Hgamma / (2. + Hzetady * dif.dt) * (bef->va.Vx[i][j + 1][k] - bef->va.Vx[i][j][k]) / dif.dy;
      }
    }
  }
#pragma omp parallel for private(i, j)
  for (j = 0; j <= Txyjmax; j++) {
    for (i = 0; i <= Txyimax; i++) {
      aft->ta.Txyx[i][j][0] = 0.;
      aft->ta.Txyx[i][j][Txykmax] = 0.;
      aft->ta.Txyy[i][j][0] = 0.;
      aft->ta.Txyy[i][j][Txykmax] = 0.;
    }
  }
//全方向加算
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Txykmax; k++) {
    for (j = 0; j <= Txyjmax; j++) {
      for (i = 0; i <= Txyimax; i++) {
        aft->ta.Txy[i][j][k] = aft->ta.Txyx[i][j][k] + aft->ta.Txyy[i][j][k];
      }
    }
  }
}

void Tyz(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, TauRan tr) {
  int i, j, k;
  int Tyzimax = tr.Tyz.x, Tyzjmax = tr.Tyz.y, Tyzkmax = tr.Tyz.z;
  double Hzetadx, Hzetady, Hzetadz, Hmu, Hgamma;
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Tyzkmax; k++) {
    for (j = 0; j <= Tyzjmax; j++) {
      for (i = 1; i <= Tyzimax - 1; i++) {
        Hzetady = 4. * pow((1. / ma.zetady[i][j + 1][k + 1]) + (1. / ma.zetady[i][j + 1][k]) + (1. / ma.zetady[i][j][k + 1]) + (1. / ma.zetady[i][j][k]), -1.);
        Hzetadz = 4. * pow((1. / ma.zetadz[i][j + 1][k + 1]) + (1. / ma.zetadz[i][j + 1][k]) + (1. / ma.zetadz[i][j][k + 1]) + (1. / ma.zetadz[i][j][k]), -1.);
        Hmu     = 4. * pow((1. /     ma.mu[i][j + 1][k + 1]) + (1. /     ma.mu[i][j + 1][k]) + (1. /     ma.mu[i][j][k + 1]) + (1. /     ma.mu[i][j][k]), -1.);
        Hgamma  = 4. * pow((1. /  ma.gamma[i][j + 1][k + 1]) + (1. /  ma.gamma[i][j + 1][k]) + (1. /  ma.gamma[i][j][k + 1]) + (1. /  ma.gamma[i][j][k]), -1.);
        aft->ta.Tyzy[i][j][k] = (2. - Hzetady * dif.dt) / (2. + Hzetady * dif.dt) * bef->ta.Tyzy[i][j][k]
         + 2. * (Hmu * dif.dt + Hgamma) / (2. + Hzetady * dif.dt) * (aft->va.Vz[i][j + 1][k] - aft->va.Vz[i][j][k]) / dif.dy
          - 2. * Hgamma / (2. + Hzetady * dif.dt) * (bef->va.Vz[i][j + 1][k] - bef->va.Vz[i][j][k]) / dif.dy;

        aft->ta.Tyzz[i][j][k] = (2. - Hzetadz * dif.dt) / (2. + Hzetadz * dif.dt) * bef->ta.Tyzz[i][j][k]
         + 2. * (Hmu * dif.dt + Hgamma) / (2. + Hzetadz * dif.dt) * (aft->va.Vy[i][j][k + 1] - aft->va.Vy[i][j][k]) / dif.dz 
          - 2. * Hgamma / (2. + Hzetadz * dif.dt) * (bef->va.Vy[i][j][k + 1] - bef->va.Vy[i][j][k]) / dif.dz;
      }
    }
  }
#pragma omp parallel for private(j, k)
  for (k = 0; k <= Tyzkmax; k++) {
    for (j = 0; j <= Tyzjmax; j++) {
      aft->ta.Tyzy[0][j][k] = 0.;
      aft->ta.Tyzy[Tyzimax][j][k] = 0.;
      aft->ta.Tyzz[0][j][k] = 0.;
      aft->ta.Tyzz[Tyzimax][j][k] = 0.;
    }
  }
//全方向加算
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Tyzkmax; k++) {
    for (j = 0; j <= Tyzjmax; j++) {
      for (i = 0; i <= Tyzimax; i++) {
        aft->ta.Tyz[i][j][k] = aft->ta.Tyzy[i][j][k] + aft->ta.Tyzz[i][j][k];
      }
    }
  }
}

void Tzx(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, TauRan tr) {
  int i, j, k;
  int Tzximax = tr.Tzx.x, Tzxjmax = tr.Tzx.y, Tzxkmax = tr.Tzx.z;
  double Hzetadx, Hzetady, Hzetadz, Hmu, Hgamma;
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Tzxkmax; k++) {
    for (j = 1; j <= Tzxjmax - 1; j++) {
      for (i = 0; i <= Tzximax; i++) {
        Hzetadx = 4. * pow((1. / ma.zetadx[i + 1][j][k + 1]) + (1. / ma.zetadx[i + 1][j][k]) + (1. / ma.zetadx[i][j][k + 1]) + (1. / ma.zetadx[i][j][k]), -1.);
        Hzetadz = 4. * pow((1. / ma.zetadz[i + 1][j][k + 1]) + (1. / ma.zetadz[i + 1][j][k]) + (1. / ma.zetadz[i][j][k + 1]) + (1. / ma.zetadz[i][j][k]), -1.);
        Hmu     = 4. * pow((1. /     ma.mu[i + 1][j][k + 1]) + (1. /     ma.mu[i + 1][j][k]) + (1. /     ma.mu[i][j][k + 1]) + (1. /     ma.mu[i][j][k]), -1.);
        Hgamma  = 4. * pow((1. /  ma.gamma[i + 1][j][k + 1]) + (1. /  ma.gamma[i + 1][j][k]) + (1. /  ma.gamma[i][j][k + 1]) + (1. /  ma.gamma[i][j][k]), -1.);
        aft->ta.Tzxz[i][j][k] = (2. - Hzetadz * dif.dt) / (2. + Hzetadz * dif.dt) * bef->ta.Tzxz[i][j][k]
         + 2. * (Hmu * dif.dt + Hgamma) / (2. + Hzetadz * dif.dt) * (aft->va.Vx[i][j][k + 1] - aft->va.Vx[i][j][k]) / dif.dz
          - 2. * Hgamma / (2. + Hzetadz * dif.dt) * (bef->va.Vx[i][j][k + 1] - bef->va.Vx[i][j][k]) / dif.dz;

        aft->ta.Tzxx[i][j][k] = (2. - Hzetadx * dif.dt) / (2. + Hzetadx * dif.dt) * bef->ta.Tzxx[i][j][k]
         + 2. * (Hmu * dif.dt + Hgamma) / (2. + Hzetadx * dif.dt) * (aft->va.Vz[i + 1][j][k] - aft->va.Vz[i][j][k]) / dif.dx
          - 2. * Hgamma / (2. + Hzetadx * dif.dt) * (bef->va.Vz[i + 1][j][k] - bef->va.Vz[i][j][k]) / dif.dx;
      }
    }
  }
#pragma omp parallel for private(i, k)
  for (k = 0; k <= Tzxkmax; k++) {
    for (i = 0; i <= Tzximax; i++) {
      aft->ta.Tzxx[i][0][k] = 0.;
      aft->ta.Tzxx[i][Tzxjmax][k] = 0.;
      aft->ta.Tzxz[i][0][k] = 0.;
      aft->ta.Tzxz[i][Tzxjmax][k] = 0.;
    }
  }
//全方向加算
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Tzxkmax; k++) {
    for (j = 0; j <= Tzxjmax; j++) {
      for (i = 0; i <= Tzximax; i++) {
        aft->ta.Tzx[i][j][k] = aft->ta.Tzxx[i][j][k] + aft->ta.Tzxz[i][j][k];
      }
    }
  }
}
//せん断応力計算
void Tau(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, TauRan tr) {
  Txy(aft, bef, ma, dif, tr);
  Tyz(aft, bef, ma, dif, tr);
  Tzx(aft, bef, ma, dif, tr);
}

void Vx(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, VelRan vr) {
  int i, j, k;
  int Vximax = vr.Vx.x, Vxjmax = vr.Vx.y, Vxkmax = vr.Vx.z;
  double Azetaxx, Azetaxy, Azetaxz, Arho;
#pragma omp parallel for private(i, j, k)
  for (k = 1; k <= Vxkmax - 1; k++) {
    for (j = 1; j <= Vxjmax - 1; j++) {
      for (i = 0; i <= Vximax; i++) {
        Azetaxx = (ma.zetaxx[i + 1][j][k] + ma.zetaxx[i][j][k]) / 2.;
        Azetaxy = (ma.zetaxy[i + 1][j][k] + ma.zetaxy[i][j][k]) / 2.;
        Azetaxz = (ma.zetaxz[i + 1][j][k] + ma.zetaxz[i][j][k]) / 2.;
        Arho    = (   ma.rho[i + 1][j][k] +    ma.rho[i][j][k]) / 2.;
        aft->va.Vxx[i][j][k] = (2. * Arho - Azetaxx * dif.dt) / (2. * Arho + Azetaxx * dif.dt) * bef->va.Vxx[i][j][k]
         + 2. * dif.dt / (2. * Arho + Azetaxx * dif.dt) * (bef->sa.Txx[i + 1][j][k] - bef->sa.Txx[i][j][k]) / dif.dx;

        aft->va.Vxy[i][j][k] = (2. * Arho - Azetaxy * dif.dt) / (2. * Arho + Azetaxy * dif.dt) * bef->va.Vxy[i][j][k]
         + 2. * dif.dt / (2. * Arho + Azetaxy * dif.dt) * (bef->ta.Txy[i][j][k] - bef->ta.Txy[i][j - 1][k]) / dif.dy;

        aft->va.Vxz[i][j][k] = (2. * Arho - Azetaxz * dif.dt) / (2. * Arho + Azetaxz * dif.dt) * bef->va.Vxz[i][j][k]
         + 2. * dif.dt / (2. * Arho + Azetaxz * dif.dt) * (bef->ta.Tzx[i][j][k] - bef->ta.Tzx[i][j][k - 1]) / dif.dz;
      }
    }
  }
#pragma omp parallel for private(i, k)
  for (k = 0; k <= Vxkmax; k++) {
    for (i = 0; i <= Vximax; i++) {
      aft->va.Vxx[i][0][k] = 0.;
      aft->va.Vxx[i][Vxjmax][k] = 0.;
      aft->va.Vxy[i][0][k] = 0.;
      aft->va.Vxy[i][Vxjmax][k] = 0.;
      aft->va.Vxz[i][0][k] = 0.;
      aft->va.Vxz[i][Vxjmax][k] = 0.;
    }
  }
#pragma omp parallel for private(i, j)
  for (j = 1; j <= Vxjmax - 1; j++) {
    for (i = 0; i <= Vximax; i++) {
      aft->va.Vxx[i][j][0] = 0.;
      aft->va.Vxx[i][j][Vxkmax] = 0.;
      aft->va.Vxy[i][j][0] = 0.;
      aft->va.Vxy[i][j][Vxkmax] = 0.;
      aft->va.Vxz[i][j][0] = 0.;
      aft->va.Vxz[i][j][Vxkmax] = 0.;
    }
  }
//全方向加算
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Vxkmax; k++) {
    for (j = 0; j <= Vxjmax; j++) {
      for (i = 0; i <= Vximax; i++) {
        aft->va.Vx[i][j][k] = aft->va.Vxx[i][j][k] + aft->va.Vxy[i][j][k] + aft->va.Vxz[i][j][k];
      }
    }
  }
}

void Vy(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, VelRan vr) {
  int i, j, k;
  int Vyimax = vr.Vy.x, Vyjmax = vr.Vy.y, Vykmax = vr.Vy.z;
  double Azetayx, Azetayy, Azetayz, Arho;
#pragma omp parallel for private(i, j, k)
  for (k = 1; k <= Vykmax - 1; k++) {
    for (j = 0; j <= Vyjmax; j++) {
      for (i = 1; i <= Vyimax - 1; i++) {
        Azetayx = (ma.zetayx[i][j + 1][k] + ma.zetayx[i][j][k]) / 2.;
        Azetayy = (ma.zetayy[i][j + 1][k] + ma.zetayy[i][j][k]) / 2.;
        Azetayz = (ma.zetayz[i][j + 1][k] + ma.zetayz[i][j][k]) / 2.;
        Arho    = (   ma.rho[i][j + 1][k] +    ma.rho[i][j][k]) / 2.;
        aft->va.Vyx[i][j][k] = (2. * Arho - Azetayx * dif.dt) / (2. * Arho + Azetayx * dif.dt) * bef->va.Vyx[i][j][k]
         + 2. * dif.dt / (2. * Arho + Azetayx * dif.dt) * (bef->ta.Txy[i][j][k] - bef->ta.Txy[i - 1][j][k]) / dif.dx;
        aft->va.Vyy[i][j][k] = (2. * Arho - Azetayy * dif.dt) / (2. * Arho + Azetayy * dif.dt) * bef->va.Vyy[i][j][k]
         + 2. * dif.dt / (2. * Arho + Azetayy * dif.dt) * (bef->sa.Tyy[i][j + 1][k] - bef->sa.Tyy[i][j][k]) / dif.dy;
        aft->va.Vyz[i][j][k] = (2. * Arho - Azetayz * dif.dt) / (2. * Arho + Azetayz * dif.dt) * bef->va.Vyz[i][j][k]
         + 2. * dif.dt / (2. * Arho + Azetayz * dif.dt) * (bef->ta.Tyz[i][j][k] - bef->ta.Tyz[i][j][k - 1]) / dif.dz;
      }
    }
  }
#pragma omp parallel for private(j, k)
  for (k = 0; k <= Vykmax; k++) {
    for (j = 0; j <= Vyjmax; j++) {
      aft->va.Vyx[0][j][k] = 0.;
      aft->va.Vyx[Vyimax][j][k] = 0.;
      aft->va.Vyy[0][j][k] = 0.;
      aft->va.Vyy[Vyimax][j][k] = 0.;
      aft->va.Vyz[0][j][k] = 0.;
      aft->va.Vyz[Vyimax][j][k] = 0.;
    }
  }
#pragma omp parallel for private(i, j)
  for (j = 0; j <= Vyjmax; j++) {
    for (i = 1; i <= Vyimax - 1; i++) {
      aft->va.Vyx[i][j][0] = 0.;
      aft->va.Vyx[i][j][Vykmax] = 0.;
      aft->va.Vyy[i][j][0] = 0.;
      aft->va.Vyy[i][j][Vykmax] = 0.;
      aft->va.Vyz[i][j][0] = 0.;
      aft->va.Vyz[i][j][Vykmax] = 0.;
    }
  }
//全方向加算
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Vykmax; k++) {
    for (j = 0; j <= Vyjmax; j++) {
      for (i = 0; i <= Vyimax; i++) {
        aft->va.Vy[i][j][k] = aft->va.Vyx[i][j][k] + aft->va.Vyy[i][j][k] + aft->va.Vyz[i][j][k];
      }
    }
  }
}

void Vz(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, VelRan vr) {
  int i, j, k;
  int Vzimax = vr.Vz.x, Vzjmax = vr.Vz.y, Vzkmax = vr.Vz.z;
  double Azetazx, Azetazy, Azetazz, Arho;
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Vzkmax; k++) {
    for (j = 1; j <= Vzjmax - 1; j++) {
      for (i = 1; i <= Vzimax - 1; i++) {
        Azetazx = (ma.zetazx[i][j][k + 1] + ma.zetazx[i][j][k]) / 2.;
        Azetazy = (ma.zetazy[i][j][k + 1] + ma.zetazy[i][j][k]) / 2.;
        Azetazz = (ma.zetazz[i][j][k + 1] + ma.zetazz[i][j][k]) / 2.;
        Arho    = (   ma.rho[i][j][k + 1] +    ma.rho[i][j][k]) / 2.;
        aft->va.Vzx[i][j][k] = (2. * Arho - Azetazx * dif.dt) / (2. * Arho + Azetazx * dif.dt) * bef->va.Vzx[i][j][k]
         + 2. * dif.dt / (2. * Arho + Azetazx * dif.dt) * (bef->ta.Tzx[i][j][k] - bef->ta.Tzx[i - 1][j][k]) / dif.dx;
        aft->va.Vzy[i][j][k] = (2. * Arho - Azetazy * dif.dt) / (2. * Arho + Azetazy * dif.dt) * bef->va.Vzy[i][j][k]
         + 2. * dif.dt / (2. * Arho + Azetazy * dif.dt) * (bef->ta.Tyz[i][j][k] - bef->ta.Tyz[i][j - 1][k]) / dif.dy;
        aft->va.Vzz[i][j][k] = (2. * Arho - Azetazz * dif.dt) / (2. * Arho + Azetazz * dif.dt) * bef->va.Vzz[i][j][k]
         + 2. * dif.dt / (2. * Arho + Azetazz * dif.dt) * (bef->sa.Tzz[i][j][k + 1] - bef->sa.Tzz[i][j][k]) / dif.dz;
      }
    }
  }
#pragma omp parallel for private(j, k)
  for (k = 0; k <= Vzkmax; k++) {
    for (j = 0; j <= Vzjmax; j++) {
      aft->va.Vzx[0][j][k] = 0.;
      aft->va.Vzx[Vzimax][j][k] = 0.;
      aft->va.Vzy[0][j][k] = 0.;
      aft->va.Vzy[Vzimax][j][k] = 0.;
      aft->va.Vzz[0][j][k] = 0.;
      aft->va.Vzz[Vzimax][j][k] = 0.;
    }
  }
#pragma omp parallel for private(i, k)
  for (k = 0; k <= Vzkmax; k++) {
    for (i = 1; i <= Vzimax - 1; i++) {
      aft->va.Vzx[i][0][k] = 0.;
      aft->va.Vzx[i][Vzjmax][k] = 0.;
      aft->va.Vzy[i][0][k] = 0.;
      aft->va.Vzy[i][Vzjmax][k] = 0.;
      aft->va.Vzz[i][0][k] = 0.;
      aft->va.Vzz[i][Vzjmax][k] = 0.;
    }
  }
//全方向加算
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Vzkmax; k++) {
    for (j = 0; j <= Vzjmax; j++) {
      for (i = 0; i <= Vzimax; i++) {
        aft->va.Vz[i][j][k] = aft->va.Vzx[i][j][k] + aft->va.Vzy[i][j][k] + aft->va.Vzz[i][j][k];
      }
    }
  }
}
//粒子速度計算
void Vel(BefAft *aft, BefAft *bef, MedArr ma, Diff dif, VelRan vr) {
  Vx(aft, bef, ma, dif, vr);
  Vy(aft, bef, ma, dif, vr);
  Vz(aft, bef, ma, dif, vr);
}

void Acc(Coord_acc *A,BefAft *aft, BefAft *bef, Diff dif, Coord out, Coord center){
  A->x = ((aft->va.Vx[out.x - 1][out.y][out.z] - bef->va.Vx[out.x - 1][out.y][out.z]) / dif.dt  + (aft->va.Vx[out.x][out.y][out.z] - bef->va.Vx[out.x][out.y][out.z]) / dif.dt) / 2;
  A->y = ((aft->va.Vy[out.x][out.y - 1][out.z] - bef->va.Vy[out.x][out.y - 1][out.z]) / dif.dt  + (aft->va.Vy[out.x][out.y][out.z] - bef->va.Vy[out.x][out.y][out.z]) / dif.dt) / 2;
  A->z = ((aft->va.Vz[out.x][out.y][out.z - 1] - bef->va.Vz[out.x][out.y][out.z - 1]) / dif.dt  + (aft->va.Vz[out.x][out.y][out.z] - bef->va.Vz[out.x][out.y][out.z]) / dif.dt) / 2;

}
//更新
void swapBefAft(BefAft *aft, BefAft *bef, Range ran) {
  int i, j, k;
  int Txximax = ran.sr.Txx.x, Txxjmax = ran.sr.Txx.y, Txxkmax = ran.sr.Txx.z;
  int Tyyimax = ran.sr.Tyy.x, Tyyjmax = ran.sr.Tyy.y, Tyykmax = ran.sr.Tyy.z;
  int Tzzimax = ran.sr.Tzz.x, Tzzjmax = ran.sr.Tzz.y, Tzzkmax = ran.sr.Tzz.z;
  int Txyimax = ran.tr.Txy.x, Txyjmax = ran.tr.Txy.y, Txykmax = ran.tr.Txy.z;
  int Tyzimax = ran.tr.Tyz.x, Tyzjmax = ran.tr.Tyz.y, Tyzkmax = ran.tr.Tyz.z;
  int Tzximax = ran.tr.Tzx.x, Tzxjmax = ran.tr.Tzx.y, Tzxkmax = ran.tr.Tzx.z;
  int Vximax = ran.vr.Vx.x, Vxjmax = ran.vr.Vx.y, Vxkmax = ran.vr.Vx.z;
  int Vyimax = ran.vr.Vy.x, Vyjmax = ran.vr.Vy.y, Vykmax = ran.vr.Vy.z;
  int Vzimax = ran.vr.Vz.x, Vzjmax = ran.vr.Vz.y, Vzkmax = ran.vr.Vz.z;
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Txxkmax; k++) {
    for (j = 0; j <= Txxjmax; j++) {
      for (i = 0; i <= Txximax; i++) {
        bef->sa.Txx[i][j][k] = aft->sa.Txx[i][j][k];
        bef->sa.Txxx[i][j][k] = aft->sa.Txxx[i][j][k];
        bef->sa.Txxy[i][j][k] = aft->sa.Txxy[i][j][k];
        bef->sa.Txxz[i][j][k] = aft->sa.Txxz[i][j][k];
      }
    }
  }
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Tyykmax; k++) {
    for (j = 0; j <= Tyyjmax; j++) {
      for (i = 0; i <= Tyyimax; i++) {
        bef->sa.Tyy[i][j][k] = aft->sa.Tyy[i][j][k];
        bef->sa.Tyyx[i][j][k] = aft->sa.Tyyx[i][j][k];
        bef->sa.Tyyy[i][j][k] = aft->sa.Tyyy[i][j][k];
        bef->sa.Tyyz[i][j][k] = aft->sa.Tyyz[i][j][k];
      }
    }
  }
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Tzzkmax; k++) {
    for (j = 0; j <= Tzzjmax; j++) {
      for (i = 0; i <= Tzzimax; i++) {
        bef->sa.Tzz[i][j][k] = aft->sa.Tzz[i][j][k];
        bef->sa.Tzzx[i][j][k] = aft->sa.Tzzx[i][j][k];
        bef->sa.Tzzy[i][j][k] = aft->sa.Tzzy[i][j][k];
        bef->sa.Tzzz[i][j][k] = aft->sa.Tzzz[i][j][k];
      }
    }
  }
#pragma omp parallel for private(i, j, k)
  for (i = 0; i <= Txyimax; i++) {
    for (j = 0; j <= Txyjmax; j++) {
      for (k = 0; k <= Txykmax; k++) {
        bef->ta.Txy[i][j][k] = aft->ta.Txy[i][j][k];
        bef->ta.Txyx[i][j][k] = aft->ta.Txyx[i][j][k];
        bef->ta.Txyy[i][j][k] = aft->ta.Txyy[i][j][k];
      }
    }
  }
#pragma omp parallel for private(i, j, k)
  for (i = 0; i <= Tyzimax; i++) {
    for (j = 0; j <= Tyzjmax; j++) {
      for (k = 0; k <= Tyzkmax; k++) {
        bef->ta.Tyz[i][j][k] = aft->ta.Tyz[i][j][k];
        bef->ta.Tyzy[i][j][k] = aft->ta.Tyzy[i][j][k];
        bef->ta.Tyzz[i][j][k] = aft->ta.Tyzz[i][j][k];
      }
    }
  }
#pragma omp parallel for private(i, j, k)
  for (i = 0; i <= Tzximax; i++) {
    for (j = 0; j <= Tzxjmax; j++) {
      for (k = 0; k <= Tzxkmax; k++) {
        bef->ta.Tzx[i][j][k] = aft->ta.Tzx[i][j][k];
        bef->ta.Tzxz[i][j][k] = aft->ta.Tzxz[i][j][k];
        bef->ta.Tzxx[i][j][k] = aft->ta.Tzxx[i][j][k];
      }
    }
  }
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Vxkmax; k++) {
    for (j = 0; j <= Vxjmax; j++) {
      for (i = 0; i <= Vximax; i++) {
        bef->va.Vx[i][j][k] = aft->va.Vx[i][j][k];
        bef->va.Vxx[i][j][k] = aft->va.Vxx[i][j][k];
        bef->va.Vxy[i][j][k] = aft->va.Vxy[i][j][k];
        bef->va.Vxz[i][j][k] = aft->va.Vxz[i][j][k];
      }
    }
  }
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Vykmax; k++) {
    for (j = 0; j <= Vyjmax; j++) {
      for (i = 0; i <= Vyimax; i++) {
        bef->va.Vy[i][j][k] = aft->va.Vy[i][j][k];
        bef->va.Vyx[i][j][k] = aft->va.Vyx[i][j][k];
        bef->va.Vyy[i][j][k] = aft->va.Vyy[i][j][k];
        bef->va.Vyz[i][j][k] = aft->va.Vyz[i][j][k];
      }
    }
  }
#pragma omp parallel for private(i, j, k)
  for (k = 0; k <= Vzkmax; k++) {
    for (j = 0; j <= Vzjmax; j++) {
      for (i = 0; i <= Vzimax; i++) {
        bef->va.Vz[i][j][k] = aft->va.Vz[i][j][k];
        bef->va.Vzx[i][j][k] = aft->va.Vzx[i][j][k];
        bef->va.Vzy[i][j][k] = aft->va.Vzy[i][j][k];
        bef->va.Vzz[i][j][k] = aft->va.Vzz[i][j][k];
      }
    }
  }
}