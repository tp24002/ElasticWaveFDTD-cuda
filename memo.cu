void MemoryBefAftToDevice(BefAft *h, BefAft *d, Range ran) {
    hipMemcpy(&h, &d, sizeof(BefAft), hipMemcpyDeviceToHost);
    hipMemcpy(&h->sa, &d->sa, sizeof(SigArr), hipMemcpyDeviceToHost);
    hipMemcpy(&h->ta, &d->ta, sizeof(TauArr), hipMemcpyDeviceToHost);
    hipMemcpy(&h->va, &d->va, sizeof(VelArr), hipMemcpyDeviceToHost);
    for(int i = 0; i < ran.sr.Txx.x; i++) {
        for(int j = 0; j < ran.sr.Txx.y; j++){
            hipMemcpy(&h->sa.Txx, &d->sa.Txx, sizeof(double) * ran.sr.Txx.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->sa.Txxx, &d->sa.Txxx, sizeof(double) * ran.sr.Txx.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->sa.Txxy, &d->sa.Txxy, sizeof(double) * ran.sr.Txx.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->sa.Txxz, &d->sa.Txxz, sizeof(double) * ran.sr.Txx.z, hipMemcpyDeviceToHost);
        }
    }
    for(int i = 0; i < ran.sr.Tyy.x; i++) {
        for(int j = 0; j < ran.sr.Tyy.y; j++){
            hipMemcpy(&h->sa.Tyy[i][j], &d->sa.Tyy[i][j], sizeof(double) * ran.sr.Tyy.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->sa.Tyyx[i][j], &d->sa.Tyyx[i][j], sizeof(double) * ran.sr.Tyy.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->sa.Tyyy[i][j], &d->sa.Tyyy[i][j], sizeof(double) * ran.sr.Tyy.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->sa.Tyyz[i][j], &d->sa.Tyyz[i][j], sizeof(double) * ran.sr.Tyy.z, hipMemcpyDeviceToHost);
        }
    }
    for(int i = 0; i < ran.sr.Tzz.x; i++) {
        for(int j = 0; j < ran.sr.Tzz.y; j++){
            hipMemcpy(&h->sa.Tzz[i][j], &d->sa.Tzz[i][j], sizeof(double) * ran.sr.Tzz.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->sa.Tzzx[i][j], &d->sa.Tzzx[i][j], sizeof(double) * ran.sr.Tzz.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->sa.Tzzy[i][j], &d->sa.Tzzy[i][j], sizeof(double) * ran.sr.Tzz.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->sa.Tzzz[i][j], &d->sa.Tzzz[i][j], sizeof(double) * ran.sr.Tzz.z, hipMemcpyDeviceToHost);
        }
    }
    //////
    for(int i = 0; i < ran.tr.Txy.x; i++) {
        for(int j = 0; j < ran.tr.Txy.y; j++){
            hipMemcpy(&h->ta.Txy[i][j], &d->ta.Txy[i][j], sizeof(double) * ran.tr.Txy.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->ta.Txyx[i][j], &d->ta.Txyx[i][j], sizeof(double) * ran.tr.Txy.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->ta.Txyy[i][j], &d->ta.Txyy[i][j], sizeof(double) * ran.tr.Txy.z, hipMemcpyDeviceToHost);
        }
    }
    for(int i = 0; i < ran.tr.Tyz.x; i++) {
        for(int j = 0; j < ran.tr.Tyz.y; j++){
            hipMemcpy(&h->ta.Tyz[i][j], &d->ta.Tyz[i][j], sizeof(double) * ran.tr.Tyz.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->ta.Tyzy[i][j], &d->ta.Tyzy[i][j], sizeof(double) * ran.tr.Tyz.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->ta.Tyzz[i][j], &d->ta.Tyzz[i][j], sizeof(double) * ran.tr.Tyz.z, hipMemcpyDeviceToHost);
        }
    }
    for(int i = 0; i < ran.tr.Tzx.x; i++) {
        for(int j = 0; j < ran.tr.Tzx.y; j++){
            hipMemcpy(&h->ta.Tzx[i][j], &d->ta.Tzx[i][j], sizeof(double) * ran.tr.Tzx.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->ta.Tzxz[i][j], &d->ta.Tzxz[i][j], sizeof(double) * ran.tr.Tzx.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->ta.Tzxx[i][j], &d->ta.Tzxx[i][j], sizeof(double) * ran.tr.Tzx.z, hipMemcpyDeviceToHost);
        }
    }
    ///////
    for(int i = 0; i < ran.vr.Vx.x; i++) {
        for(int j = 0; j < ran.vr.Vx.y; j++){
            hipMemcpy(&h->va.Vx, &d->va.Vx, sizeof(double) * ran.vr.Vx.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->va.Vxx, &d->va.Vxx, sizeof(double) * ran.vr.Vx.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->va.Vxy, &d->va.Vxy, sizeof(double) * ran.vr.Vx.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->va.Vxz, &d->va.Vxz, sizeof(double) * ran.vr.Vx.z, hipMemcpyDeviceToHost);
        }
    }
    for(int i = 0; i < ran.vr.Vy.x; i++) {
        for(int j = 0; j < ran.vr.Vy.y; j++){
            hipMemcpy(&h->va.Vy, &d->va.Vy, sizeof(double) * ran.vr.Vy.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->va.Vyx, &d->va.Vyx, sizeof(double) * ran.vr.Vy.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->va.Vyy, &d->va.Vyy, sizeof(double) * ran.vr.Vy.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->va.Vyz, &d->va.Vyz, sizeof(double) * ran.vr.Vy.z, hipMemcpyDeviceToHost);
        }
    }
    for(int i = 0; i < ran.vr.Vz.x; i++) {
        for(int j = 0; j < ran.vr.Vz.y; j++) {
            hipMemcpy(&h->va.Vz, &d->va.Vz, sizeof(double) * ran.vr.Vz.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->va.Vzx, &d->va.Vzx, sizeof(double) * ran.vr.Vz.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->va.Vzy, &d->va.Vzy, sizeof(double) * ran.vr.Vz.z, hipMemcpyDeviceToHost);
            hipMemcpy(&h->va.Vzz, &d->va.Vzz, sizeof(double) * ran.vr.Vz.z, hipMemcpyDeviceToHost);
        }
    }
}