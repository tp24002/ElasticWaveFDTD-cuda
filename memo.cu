    hipMemcpy(&d_medarrptr->ramda, &h_medarrptr->ramda, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->mu, &h_medarrptr->mu, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->c11, &h_medarrptr->c11, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->rho, &h_medarrptr->rho, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->zetaxx, &h_medarrptr->zetaxx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetaxy, &h_medarrptr->zetaxy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetaxz, &h_medarrptr->zetaxz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->zetayx, &h_medarrptr->zetayx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetayy, &h_medarrptr->zetayy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetayz, &h_medarrptr->zetayz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->zetazx, &h_medarrptr->zetazx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetazy, &h_medarrptr->zetazy, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetazz, &h_medarrptr->zetazz, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->gamma, &h_medarrptr->gamma, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->khi, &h_medarrptr->khi, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->xi11, &h_medarrptr->xi11, sizeof(double*), hipMemcpyHostToDevice);

    hipMemcpy(&d_medarrptr->zetadx, &h_medarrptr->zetadx, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetady, &h_medarrptr->zetady, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&d_medarrptr->zetadz, &h_medarrptr->zetadz, sizeof(double*), hipMemcpyHostToDevice);