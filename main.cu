#include "hip/hip_runtime.h"
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "./header/init.h"
#include "./header/struct.h"
#include "./header/update.h"
#include "./header/parameter.h"
#include "./header/memory.h"

void progressBar(int now, int max);
void AccelerationCalculation(AccCoord *Acc, BefAft aft, BefAft bef, Diff dif, Coord out, Range ran);

int main(void) {
  // 静的変数
  Medium med_h[E_M_END];
  Object air_h;
  Object con_h;
  Object clack_h;
  Range ran_h, *ran_d;
  Pml pml_h;
  Diff dif_h, *dif_d;
  int tmax_h;
  int t_h;
  int outNum_h;


  // 動的変数(計算領域の大きさで大きさ決定)
  MedArr *ma_h, *ma_d;
  BefAft bef_h, *bef_d;
  BefAft aft_h, *aft_d;
  Impulse *ip_h, *ip_d;
  
  AccCoord *acc_h;

  // int RegionArea;

  FILE *fp1;
  // FILE *fp1,*fp2,*fp3,*fp4;
  char fn1[256];
  // char fn1[256],fn2[256],fn3[256],fn4[256];
  // int tmp = 0;

  Coord *out_h;
  // Coord center;
  // // int make_models; // 作成するモデルの数
  // int model_count = 0; // いくつ目のモデルを作成中か
  // int ratio;
  // int max_Patern; // コンクリートのセル数
  // // int max_ClackPatern; // 欠陥を配置できる最大のパターン数
  // int clack_count; // 割合による欠陥数
  Coord threads;

  // スレッド数
  initCoord(&threads, 4, 4, 8);
  
  // データ格納
  StaticVariable(med_h, &pml_h, &ran_h, &dif_h, &air_h, &con_h, &clack_h, &tmax_h, &outNum_h);
  // ホスト動的変数
  ma_h  = allocateHostMedArr(&ran_h);
  ip_h  = allocateHostImpulse(&ran_h);
  acc_h = allocateHostAccCoord(outNum_h);
  out_h = allocateHostCoord(outNum_h);
  
  // デバイス動的変数
  ma_d  = allocateDeviceMedArr(&ran_h);
  bef_d = allocateDeviceBefAft(&ran_h);
  aft_d = allocateDeviceBefAft(&ran_h);
  ip_d  = allocateDeviceImpulse(&ran_h);

  // デバイス静的変数
  hipMalloc(&ran_d, sizeof(Range));
  hipMalloc(&dif_d, sizeof(Diff));

  DynamicVariable(acc_h, ma_h, ip_h, ran_h, air_h, con_h, clack_h, pml_h, out_h, outNum_h);

  // ホスト->デバイス　データ転送
  RangeHostToDevice(&ran_h, ran_d);
  DiffHostToDevice(&dif_h, dif_d);

  MedArrHostToDevice(ma_h, ma_d, ran_h);
  ImpulseHostToDevice(ip_h, ip_d, ran_h);

  dim3 threadsPerBlock(threads.x, threads.y, threads.z);  // ブロック内のスレッド数
  dim3 ZeroTBlocks((ran_h.sr.Txx.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (ran_h.sr.Txx.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (ran_h.sr.Txx.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 ZeroTxyBlocks((ran_h.tr.Txy.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h.tr.Txy.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h.tr.Txy.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 ZeroTyzBlocks((ran_h.tr.Tyz.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h.tr.Tyz.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h.tr.Tyz.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 ZeroTzxBlocks((ran_h.tr.Tzx.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (ran_h.tr.Tzx.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                     (ran_h.tr.Tzx.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 ZeroVxBlocks((ran_h.vr.Vx.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (ran_h.vr.Vx.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (ran_h.vr.Vx.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 ZeroVyBlocks((ran_h.vr.Vy.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (ran_h.vr.Vy.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (ran_h.vr.Vy.z + threadsPerBlock.z - 1) / threadsPerBlock.z);
  dim3 ZeroVzBlocks((ran_h.vr.Vz.x + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (ran_h.vr.Vz.y + threadsPerBlock.y - 1) / threadsPerBlock.y,
                    (ran_h.vr.Vz.z + threadsPerBlock.z - 1) / threadsPerBlock.z);

  // 出力
  // Medium
  for(int i = 0; i < E_M_END; i++) {
    printf("Med:%f\n", med_h[i].rho);
  }
  // Pml
  printf("pml.fm:%f\n", pml_h.fm);
  printf("pml.ta:%f\n", pml_h.ta);
  // Range
  printf("Range:%d,%d,%d\n", ran_h.sr.Txx.x, ran_h.sr.Txx.y, ran_h.sr.Txx.z);
  // Diff
  printf("sp   diff:%f,%f,%f\n", dif_h.dx, dif_h.dy, dif_h.dz);
  printf("time diff:%e\n", dif_h.dt);
  // Impulse
  // printf("")
  printf("time:%d\n", tmax_h);

  ///////////clack
  // ratio = 10;
  // max_Patern = con_size.x * con_size.y * con_size.z;
  // // max_ClackPatern = (con_size.x - 2) * (con_size.y - 2) * (con_size.z - 2);
  // clack_count = max_Patern * ratio / 100;
  // if(ratio != 0){
  //   clack = (Object *)malloc(sizeof(Object) * clack_count);
  //   printf("clackhalfok\n");
  //   initClack(clack,med[E_AIR], &pml, clack_st.x, clack_st.y, clack_st.z, clack_size.x, clack_size.y, clack_size.z);
  //   printf("ratio:%d\n", ratio);
  //   insertClack(&ma_h, clack, ratio);
  // }
  // if(ratio != 0){
  //   // model_count++;
  //   sprintf(fn1, "./clack/ratio%d/clack_%d.csv", ratio, (model_count + 1));
  //   fp1 = fopen(fn1, "wb");
  //   fprintf(fp1, "sp.x,sp.y,sp.z,ln.x,ln.y,ln,z\n");
  //   // for(int i = 0; i < clack_count; i++){
  //   //   fprintf(fp1, "%d,%d,%d,", clack[i].sp.x, clack[i].sp.y, clack[i].sp.z, clack[i].range.x,clack[i].range.y, clack[i].range.z);
  //   // }
  // }

  
  // int idx, idy, idz;
  // idx = ip_h->in.z * ran_h.sr.Txx.x * ran_h.sr.Txx.y + ip_h->in.y * ran_h.sr.Txx.x + ip_h->in.x;
  // idy = out_h[0].z * ran_h.sr.Txx.x * ran_h.sr.Txx.y + out_h[0].y * ran_h.sr.Txx.x + out_h[0].x;
  // idz = out_h[1].z * ran_h.sr.Txx.x * ran_h.sr.Txx.y + out_h[1].y * ran_h.sr.Txx.x + out_h[1].x;
  // for(int i = 0; i < ran_h.sr.Txx.x; i++) {
  //   for(int j = 0; j < ran_h.sr.Txx.y; j++) {
  //     id = j * ran_h.sr.Txx.x * ran_h.sr.Txx.y + ip_h->in.y * ran_h.sr.Txx.x + i;
  //     if(id == idx) {
  //       printf("*");
  //     } else if(id == idy) {
  //       printf("x");
  //     } else if(id == idz) {
  //       printf("x");
  //     } else {
  //       printf("o");
  //     }
  //   }
  //   printf("\n");
  // }

  //ファイル名出力
  // printf("%.*s\n", (int) sizeof fn1, fn1);
  // fp1 = fopen(fn1, "wb");

  // double test;
  int ratio = 10;
  int model_count = 0;
  sprintf(fn1, "./clack/ratio%d/clack_%d.csv", ratio, (model_count + 1));
  fp1 = fopen(fn1, "w");

  // 0 padding
  ZeroT<<<ZeroTBlocks,threadsPerBlock>>>(aft_d, ran_d);
  ZeroTxy<<<ZeroTxyBlocks,threadsPerBlock>>>(aft_d, ran_d);
  ZeroTyz<<<ZeroTyzBlocks,threadsPerBlock>>>(aft_d, ran_d);
  ZeroTzx<<<ZeroTzxBlocks,threadsPerBlock>>>(aft_d, ran_d);
  ZeroVx<<<ZeroVxBlocks,threadsPerBlock>>>(aft_d, ran_d);
  ZeroVy<<<ZeroVyBlocks,threadsPerBlock>>>(aft_d, ran_d);
  ZeroVz<<<ZeroVzBlocks,threadsPerBlock>>>(aft_d, ran_d);
  for (t_h = 0; t_h < tmax_h; t_h++) {
    // 入力情報作成
    insertImpulse(ip_h, dif_h, t_h, ran_h);
    ImpulseHostToDevice(ip_h, ip_d, ran_h);

    Vel(aft_d, bef_d, ma_d, dif_d, ran_d, &ran_h, threads);
    Sig(aft_d, bef_d, ma_d, dif_d, ran_d, &ran_h, ip_d, threads);
    Tau(aft_d, bef_d, ma_d, dif_d, ran_d, &ran_h, threads);

    BefAftDeviceToHost(aft_d, &aft_h, ran_h);
    BefAftDeviceToHost(bef_d, &bef_h, ran_h);

    for(int j = 0; j < outNum_h; j++) {
      AccelerationCalculation(&acc_h[j], aft_h, bef_h, dif_h, out_h[j], ran_h);
      fprintf(fp1,"%le,%le,%le,", acc_h[j].x, acc_h[j].y, acc_h[j].z);
    }
    fprintf(fp1, "\n");

    swapBefAft(aft_d, bef_d, &ran_h, ran_d, threads);
    progressBar(t_h, tmax_h);
  }
  printf("%lf\n", aft_h.sa.Tzz[0]);
  fclose(fp1);
  printf("loop end\n");
  return 0;
}

void progressBar(int now, int max) {
  int bar_width = 50;
  double progress = (double)(now + 1) / (double)max;
  int bar_length = (int)(progress * bar_width);
  printf("Progress: [");
  for (int j = 0; j < bar_length; j++) {
    printf("=");
  }
  for (int j = bar_length; j < bar_width; j++) {
    printf(" ");
  }
  printf("] %.2f%%\r", progress * 100);
  fflush(stdout);
}


void AccelerationCalculation(AccCoord *Acc, BefAft aft, BefAft bef, Diff dif, Coord out, Range ran) {

  int xmax = ran.sr.Txx.x, ymax = ran.sr.Txx.y;

  int x = out.x;
  int y = out.y;
  int z = out.z;

  // 1Dインデックスの計算
  int idx   = z * (xmax * ymax) + y * xmax + x;
  int idxX  = z * (xmax * ymax) + y * xmax + (x + 1);
  int idxY  = z * (xmax * ymax) + (y + 1) * xmax + x;
  int idxZ  = (z + 1) * (xmax * ymax) + y * xmax + x;

  Acc->x = ((*(aft.va.Vx + idxX) - *(bef.va.Vx + idxX)) / dif.dt + (*(aft.va.Vx + idx) - *(bef.va.Vx + idx)) / dif.dt) / 2;

  Acc->y = ((*(aft.va.Vy + idxY) - *(bef.va.Vy + idxY)) / dif.dt + (*(aft.va.Vy + idx) - *(bef.va.Vy + idx)) / dif.dt) / 2;

  Acc->z = ((*(aft.va.Vz + idxZ) - *(bef.va.Vz + idxZ)) / dif.dt + (*(aft.va.Vz + idx) - *(bef.va.Vz + idx)) / dif.dt) / 2;
  // printf("acc:%f,%f,%f\n", Acc.x, Acc.y, Acc.z);
}