#include "hip/hip_runtime.h"
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "./header/init.h"
#include "./header/insert.h"
#include "./header/struct.h"
#include "./header/update.h"
#include "./header/parameter.h"
#include "./header/memory.h"

void progressBar(int now, int max);

int main(void) {
  // 宣言
  Medium med[E_M_END];
  Object *con;
  Object *clack;
  Range ran;
  Pml *pml;
  //host data(cpu)
  Diff *dif_h, *dif_d;
  MedArr *ma_h, *ma_d;
  BefAft *bef_h, *bef_d;
  BefAft *aft_h, *aft_d;
  Impulse *ip_h, *ip_d;
  int tmax;


  // Coord out1,out2,out3,out4;
  FILE *fp1;
  // FILE *fp1,*fp2,*fp3,*fp4;
  // char fn1[256];
  // char fn1[256],fn2[256],fn3[256],fn4[256];
  // int tmp = 0;

  Coord out[16];
  // Coord center;
  int outNum = 10;
  // // int make_models; // 作成するモデルの数
  // int model_count = 0; // いくつ目のモデルを作成中か
  // int ratio;
  // int max_Patern; // コンクリートのセル数
  // // int max_ClackPatern; // 欠陥を配置できる最大のパターン数
  // int clack_count; // 割合による欠陥数
  Coord threads;
  Coord_acc **Acc_h, **Acc_d;

  // スレッド数
  initCoord(&threads, 4, 4, 8);
  // ran = (Range *)malloc(sizeof(Range));
  pml = (Pml *)malloc(sizeof(Pml));
  dif_h = (Diff *)malloc(sizeof(Diff));
  con = (Object *)malloc(sizeof(Object));
  clack = (Object *)malloc(sizeof(Object));
  ma_h = (MedArr *)malloc(sizeof(MedArr));

  // データ格納
  StaticVariable(&ran, pml, dif_h, con, clack, med, ma_h, &tmax);

  allocateHostBefAft(&bef_h, ran);
  allocateHostBefAft(&aft_h, ran);
  zeroPadding(bef_h, ran);
  zeroPadding(aft_h, ran);
  printf("aaa\n");
  printf("a:%f\n", bef_h->sa.Txx[73][73][73]);
  printf("nono\n");

  allocateHostMedArr(&ma_h, ran);
  printf("nono\n");
  allocateHostImpulse(&ip_h, ran);
  DynamicVariable(ma_h, ip_h, ran, med, *con, *clack, *pml, *dif_h, tmax);
  
  // 出力
  printf("time:%d\n", tmax);
  printf("range:%d,%d,%d(in pml)\n", ran.sr.Txx.x, ran.sr.Txx.y, ran.sr.Txx.z);
  printf("pml:%d,%d,%d\n", pml->pl1.x, pml->pl1.y, pml->pl1.z);
  printf("dif_xyz:%f,%f,%f\n", dif_h->dx, dif_h->dy, dif_h->dz);
  printf("dif_time:%f\n", dif_h->dt);
  printf("in:%d,%d,%d\n", ip_h->in.x, ip_h->in.y, ip_h->in.z);
  if(ip_h->mode == E_SINE){
    printf("sin:%f\n", ip_h->freq);
  } else if(ip_h->mode == E_RCOS){
    printf("cos:%f\n", ip_h->freq);
  }

  // 関数化推奨
  // 加速度メモリ確保
  Acc_h = (Coord_acc **)malloc(sizeof(Coord_acc *) * outNum);
  for (int i = 0; i < outNum; i++) {
    Acc_h[i] = (Coord_acc *)malloc(tmax * sizeof(Coord_acc));
  }
  hipMalloc((void **)&Acc_d, outNum * sizeof(Coord_acc *));
  for (int i = 0; i < outNum; i++) {
    Coord_acc *temp_d;
    hipMalloc((void **)&temp_d, tmax * sizeof(Coord_acc));
    hipMemcpy(Acc_h[i], temp_d, tmax * sizeof(Coord_acc), hipMemcpyHostToDevice);
    hipMemcpy(&Acc_d[i], &temp_d, sizeof(Coord_acc *), hipMemcpyHostToDevice);
  }
  ///////////clack
  // ratio = 10;
  // max_Patern = con_size.x * con_size.y * con_size.z;
  // // max_ClackPatern = (con_size.x - 2) * (con_size.y - 2) * (con_size.z - 2);
  // clack_count = max_Patern * ratio / 100;
  // if(ratio != 0){
  //   clack = (Object *)malloc(sizeof(Object) * clack_count);
  //   printf("clackhalfok\n");
  //   initClack(clack,med[E_AIR], &pml, clack_st.x, clack_st.y, clack_st.z, clack_size.x, clack_size.y, clack_size.z);
  //   printf("ratio:%d\n", ratio);
  //   insertClack(&ma_h, clack, ratio);
  // }
  // if(ratio != 0){
  //   // model_count++;
  //   sprintf(fn1, "./clack/ratio%d/clack_%d.csv", ratio, (model_count + 1));
  //   fp1 = fopen(fn1, "wb");
  //   fprintf(fp1, "sp.x,sp.y,sp.z,ln.x,ln.y,ln,z\n");
  //   // for(int i = 0; i < clack_count; i++){
  //   //   fprintf(fp1, "%d,%d,%d,", clack[i].sp.x, clack[i].sp.y, clack[i].sp.z, clack[i].range.x,clack[i].range.y, clack[i].range.z);
  //   // }
  // }


  //ファイル名出力
  // printf("%.*s\n", (int) sizeof fn1, fn1);
  // fp1 = fopen(fn1, "wb");

  // device構造体本体のメモリ確保
  hipMalloc((void **)&aft_d, sizeof(BefAft));
  hipMalloc((void **)&bef_d, sizeof(BefAft));
  hipMalloc((void **)&ma_d, sizeof(MedArr));
  hipMalloc((void **)&dif_d, sizeof(Diff));
  hipMalloc((void **)&ip_d, sizeof(Impulse));
  // device構造体中身(メンバ)のメモリ確保関数
  allocateDeviceBefAft(&aft_d, ran);
  allocateDeviceBefAft(&bef_d, ran);
  printf("aloocate BefAft ok\n");
  allocateDeviceMedArr(&ma_d, ran);
  printf("aloocate MedArr ok\n");
  allocateDeviceImpulse(&ip_d, ran);
  printf("aloocate Impulse ok\n");
  // double test;
  for (int t = 0; t < tmax; t++) {

    
    // printf("%f\n",ip_h.Tzz[ip_h.in.x][ip_h.in.y][ip_h.in.z]);

    // printf("%f\n",ip_d.freq);
    Vel(aft_h, bef_h, aft_d, bef_d, *ma_h, ma_d, *dif_h, dif_d, ran, threads);
    Sig(aft_h, bef_h, aft_d, bef_d, *ma_h, ma_d, *dif_h, dif_d, ran, *ip_h, ip_d, t, threads);
    Tau(aft_h, bef_h, aft_d, bef_d, *ma_h, ma_d, *dif_h, dif_d, ran, threads);


    // 加速度算出＆書き込み
    Acceleration<<<1, 1>>>(Acc_d, aft_d, bef_d, *dif_h, out, outNum, t);

    // swapBefAft<<<1, 1>>>(&aft_d, &bef_d, ran);
    // progressBar(t, tmax);
  }

  hipError_t err = hipMemcpy(Acc_d, Acc_h, outNum * sizeof(Coord_acc *), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("acc d to h Error: %s\n", hipGetErrorString(err));
    return;
  }
  for (int j = 0; j < tmax; j++) {
    for (int i = 0; i < outNum; i++) {
      fprintf(fp1,"%le,%le,%le,", Acc_h[i][j].x, Acc_h[i][j].y, Acc_h[i][j].z);
    }
    fprintf(fp1, "\n");
    progressBar(j, tmax);
  }
  fclose(fp1);
  printf("loop end.\n");
  return 0;
}

void progressBar(int now, int max) {
  int bar_width = 50;
  double progress = (double)(now + 1) / (double)max;
  int bar_length = (int)(progress * bar_width);
  printf("Progress: [");
  for (int j = 0; j < bar_length; j++) {
    printf("=");
  }
  for (int j = bar_length; j < bar_width; j++) {
    printf(" ");
  }
  printf("] %.2f%%\r", progress * 100);
  fflush(stdout);
}
